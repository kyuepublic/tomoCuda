#include "hip/hip_runtime.h"
// Only support reflect mode right now

#include <stdio.h>
#include <medianFilter.hh>

#define IN(X,Y)  d_in[X+Y*(14+nx)]
//
//v[i++] = d_in[xx+yy*newnx+zoffset];

// various windows size
//__global__ void kernel(int nx, int ny, float *d_out, float *d_in, int size)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    int i = 0;
////    float v[9] = {0,0,0,0,0,0,0,0,0}; // zero padding
//
//    int offset = (size-1)/2;
//    int winSize = size*size;
//    int winOffset = (winSize-1)/2;
//
//    float v[winSize] = {0};
//
//
//    for (int xx = x - offset; xx <= x + offset; xx++) {
//        for (int yy = y - offset; yy <= y + offset; yy++) {
//            if (0 <= xx && xx < nx && 0 <= yy && yy < ny) // boundaries
//
//                v[i++] = d_in[yy*nx + xx];
//        }
//    }
//
//    // bubble-sort
//    for (int i = 0; i < winSize; i++) {
//        for (int j = i + 1; j < winSize; j++) {
//            if (v[i] > v[j]) { /* swap? */
//                float tmp = v[i];
//                v[i] = v[j];
//                v[j] = tmp;
//            }
//        }
//    }
//
////     printf("the x not is %d, y is %d, result is %f \n", x, y, v[4] );
//    // pick the middle one
//    d_out[y*nx + x] = v[winOffset];
//}


// window 2 by 2
__global__ void kernel2(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 2;
    float v[4] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = winSize/2 - 1;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;



    int i = 0;


    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset)
             {// boundaries
                v[i++] = d_in[yy*(nx+toffset) + xx];
             }
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];

}


// window 3 by 3
__global__ void kernel3(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 3;
    float v[9] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = (winSize-1)/2;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
}

// windows size 4 byb 4
__global__ void kernel4(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 4;
    int loffset = winSize/2;
    int roffset = winSize/2 - 1;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;
    float v[16] = {0};

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < 16; i++)
    {
        for (int j = i + 1; j < 16; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[8];
}

// Windows size 5 by b5
__global__ void kernel5(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 5;
    float v[25] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = (winSize-1)/2;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];

}

// windows size 6 byb 6
__global__ void kernel6(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 6;
    float v[36] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = winSize/2 - 1;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;


    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
}


// window size 15 by b15
__global__ void kernel15(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

        int i = 0;

        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
//                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                    v[i++] = d_in[yy*(nx+toffset) + xx];
            }
        }

        // bubble-sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                { /* swap? */
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
    }

}

__global__ void kernel3D2(int nx, int ny, int nz,  float *d_out, float *d_in)
{
   // nx ny nz map to offset in the 1d array
    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;

    if ((x < nx) && (y < ny) && (z < nz))
    {
        int winSize = 2;
        float v[4] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;
        int newnx=toffset+nx;
        int newny=toffset+ny;

        x = x + loffset;
        y = y + loffset;

        int i = 0;

        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
//                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                    v[i++] = d_in[xx+yy*newnx+z*newnx*newny];
            }
        }

        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                { /* swap? */
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];

    }

}

__global__ void kernel3D15(int nx, int ny, int nz,  float *d_out, float *d_in)
{
   // nx ny nz map to offset in the 1d array
    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;

//    int offset = x+y* nx + ny * nx * z;
    if ((x < nx) && (y < ny) && (z < nz))
    {
        // initial the window size, the local vector size
        int winSize = 15;
        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2; // the left and top offset
        int roffset = (winSize-1)/2; // the right and bottom offset
        int toffset = loffset+roffset; // the overall offset

// The new x' y' is the plus offset
        x = x + loffset;
        y = y + loffset;

        int i = 0;
        // Put the neighbour pixel into the local memory for the later bubble sort
        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
                    // find the read address of the x y z pixel
                    v[i++] = d_in[xx+yy*(nx+toffset)+z*(nx+toffset)*(ny+toffset)];
            }
        }

        // do the bubble sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                {   // bubble sort
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        //    printf("the x is %d, y is %d, z is %d, result is %f \n", x, y, z, v[vecSize/2] );
        // put the final result value to the output array
        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];

    }

}


__global__ void kernelLool3D15(int nx, int ny, int nz,  float *d_out, float *d_in)
{
   // nx ny nz map to offset in the 1d array
    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;

//    int offset = x+y* nx + ny * nx * z;
    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

        int i = 0;



        for(int z = 0; z < nz; z++)
        {
            i = 0;

            for (int xx = x - loffset; xx <= x + roffset; xx++)
            {
                for (int yy = y - loffset; yy <= y + roffset; yy++)
                {

                    v[i++] = d_in[xx+yy*(nx+toffset)+z*(nx+toffset)*(ny+toffset)];
                }
            }

            for (int i = 0; i < vecSize; i++)
            {
                for (int j = i + 1; j < vecSize; j++)
                {
                    if (v[i] > v[j])
                    { /* swap? */
                        float tmp = v[i];
                        v[i] = v[j];
                        v[j] = tmp;
                    }
                }
            }

//            printf("the x is %d, y is %d, z is %d, result is %f \n", x, y, z, v[vecSize/2] );

            d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];

        }

    }

}

__global__ void kernelLool3D15XZY(int nx, int ny, int nz,  float *d_out, float *d_in)
{
   // nx ny nz map to offset in the 1d array
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int z = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;

//    int offset = x+y* nx + ny * nx * z;
    if ((x < nx) && (z < nz))
    {
        int winSize = 15;
        float v[225]={0};


        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        int newnx = nx+toffset;
        int newny = ny+toffset;
        int zoffset = z*newnx*newny;

        x = x + loffset;
//        y = y + loffset;
//        z = z + loffset;

        int i = 0;

        for(int y = loffset; y < ny+loffset; y++)
        {
            i = 0;

//            for (int xx = x - loffset; xx <= x + roffset; xx++)
//            {
//                for (int yy = y - loffset; yy <= y + roffset; yy++)
//                {
//
//                    v[i++] = d_in[xx+yy*newnx+zoffset];
//                }
//            }



            for (int i = 0; i < vecSize; i++)
            {
                for (int j = i + 1; j < vecSize; j++)
                {
                    if (v[i] > v[j])
                    {
                        float tmp = v[i];
                        v[i] = v[j];
                        v[j] = tmp;
                    }
                }
            }

//            printf("the x is %d, y is %d, z is %d, result is %f \n", x, y, z, v[vecSize/2] );

            d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];

        }

    }

}

__global__ void reomveOutliner3D2(int nx, int ny, int nz, int diff, float *d_out, float *d_in)
{
   // nx ny nz map to offset in the 1d array
    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;

//    int offset = x+y* nx + ny * nx * z;
    if ((x < nx) && (y < ny) && (z < nz))
    {
        int winSize = 2;
        float v[4] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;
        int newnx=toffset+nx;
        int newny=toffset+ny;

        x = x + loffset;
        y = y + loffset;

        int i = 0;

        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
//                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                    v[i++] = d_in[xx+yy*newnx+z*newnx*newny];
            }
        }

        // get the current pixel value
        // TODO get from local buffer instead of global memory

        float currentPixel = d_in[x+y*newnx+z*newnx*newny];



        // More optimize for the bubble sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                { /* swap? */
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        // TODO more optimize for this part
        int mask = 0;
        float realdiff = currentPixel-v[vecSize/2];
        printf("the x is %d, y is %d, z is %d, current is %f, result is %f \n", x, y, z, currentPixel, v[vecSize/2] );

        if( realdiff >= diff)
            mask = 1;
        else
            mask = 0;



        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2]*mask+currentPixel*(1-mask);

    }

}

__global__ void reomveOutliner2D15(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

        int i = 0;

        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
                v[i++] = d_in[yy*(nx+toffset) + xx];
            }
        }

        float currentPixel = IN(x, y);

        // bubble-sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                {
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        int mask = 0;
        if((currentPixel-v[vecSize/2]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2]*mask+currentPixel*(1-mask);
    }

}

__global__ void reomveOutliner2D15M(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
//        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

//        int i = 0;

        float v[225]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7),
                    IN(x+1, y-7), IN(x+1, y-6), IN(x+1, y-5), IN(x+1, y-4), IN(x+1, y-3), IN(x+1, y-2), IN(x+1, y-1), IN(x+1, y), IN(x+1, y+1), IN(x+1, y+2), IN(x+1, y+3), IN(x+1, y+4), IN(x+1, y+5), IN(x+1, y+6), IN(x+1, y+7),
                    IN(x+2, y-7), IN(x+2, y-6), IN(x+2, y-5), IN(x+2, y-4), IN(x+2, y-3), IN(x+2, y-2), IN(x+2, y-1), IN(x+2, y), IN(x+2, y+1), IN(x+2, y+2), IN(x+2, y+3), IN(x+2, y+4), IN(x+2, y+5), IN(x+2, y+6), IN(x+2, y+7),
                    IN(x+3, y-7), IN(x+3, y-6), IN(x+3, y-5), IN(x+3, y-4), IN(x+3, y-3), IN(x+3, y-2), IN(x+3, y-1), IN(x+3, y), IN(x+3, y+1), IN(x+3, y+2), IN(x+3, y+3), IN(x+3, y+4), IN(x+3, y+5), IN(x+3, y+6), IN(x+3, y+7),
                    IN(x+4, y-7), IN(x+4, y-6), IN(x+4, y-5), IN(x+4, y-4), IN(x+4, y-3), IN(x+4, y-2), IN(x+4, y-1), IN(x+4, y), IN(x+4, y+1), IN(x+4, y+2), IN(x+4, y+3), IN(x+4, y+4), IN(x+4, y+5), IN(x+4, y+6), IN(x+4, y+7),
                    IN(x+5, y-7), IN(x+5, y-6), IN(x+5, y-5), IN(x+5, y-4), IN(x+5, y-3), IN(x+5, y-2), IN(x+5, y-1), IN(x+5, y), IN(x+5, y+1), IN(x+5, y+2), IN(x+5, y+3), IN(x+5, y+4), IN(x+5, y+5), IN(x+5, y+6), IN(x+5, y+7),
                    IN(x+6, y-7), IN(x+6, y-6), IN(x+6, y-5), IN(x+6, y-4), IN(x+6, y-3), IN(x+6, y-2), IN(x+6, y-1), IN(x+6, y), IN(x+6, y+1), IN(x+6, y+2), IN(x+6, y+3), IN(x+6, y+4), IN(x+6, y+5), IN(x+6, y+6), IN(x+6, y+7),
                    IN(x+7, y-7), IN(x+7, y-6), IN(x+7, y-5), IN(x+7, y-4), IN(x+7, y-3), IN(x+7, y-2), IN(x+7, y-1), IN(x+7, y), IN(x+7, y+1), IN(x+7, y+2), IN(x+7, y+3), IN(x+7, y+4), IN(x+7, y+5), IN(x+7, y+6), IN(x+7, y+7)
        };

        float currentPixel = IN(x, y);

        // bubble-sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                {
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        int mask = 0;
        if((currentPixel-v[vecSize/2]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2]*mask+currentPixel*(1-mask);
    }

}


__global__ void reomveOutliner3D15(int nx, int ny, int nz, int diff, float *d_out, float *d_in)
{
   // nx ny nz map to offset in the 1d array
    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;

//    int offset = x+y* nx + ny * nx * z;
    if ((x < nx) && (y < ny) && (z < nz))
    {
        int winSize = 15;
        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;
        int newnx=toffset+nx;
        int newny=toffset+ny;

        x = x + loffset;
        y = y + loffset;

        int i = 0;

        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
//                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                    v[i++] = d_in[xx+yy*newnx+z*newnx*newny];
            }
        }

        // get the current pixel value
        // TODO get from local buffer instead of global memory

        float currentPixel = d_in[x+y*newnx+z*newnx*newny];

//        printf("the x is %d, y is %d, z is %d, current is %f, result is %f \n", x, y, z, currentPixel, v[vecSize/2] );

        // More optimize for the bubble sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                { /* swap? */
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        // TODO more optimize for this part
        int mask = 0;
        if((currentPixel-v[vecSize/2]) >= diff)
            mask = 1;
        else
            mask = 0;



        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2]*mask+currentPixel*(1-mask);

    }

}


//#define s2(a,b)            { float tmp = a; a = min(a,b); b = max(tmp,b); }
//#define mn3(a,b,c)         s2(a,b); s2(a,c);
//#define mx3(a,b,c)         s2(b,c); s2(a,c);
//
//#define mnmx3(a,b,c)       mx3(a,b,c); s2(a,b);                               // 3 exchanges
//#define mnmx4(a,b,c,d)     s2(a,b); s2(c,d); s2(a,c); s2(b,d);                // 4 exchanges
//#define mnmx5(a,b,c,d,e)   s2(a,b); s2(c,d); mn3(a,c,e); mx3(b,d,e);          // 6 exchanges
//#define mnmx6(a,b,c,d,e,f) s2(a,d); s2(b,e); s2(c,f); mn3(a,b,c); mx3(d,e,f); // 7 exchanges
//
//#define SMEM(x,y)  smem[(x)+1][(y)+1]
//#define IN(x,y)    d_in[(y)*nx + (x)]
//
// __global__ void kernel(int nx, int ny, float *d_out, float *d_in, int size)
//{
//
//    int tx = threadIdx.x, ty = threadIdx.y;
//
//    // guards: is at boundary?
//    bool is_x_top = (tx == 0), is_x_bot = (tx == BLOCK_X-1);
//    bool is_y_top = (ty == 0), is_y_bot = (ty == BLOCK_Y-1);
//
//    __shared__ float smem[BLOCK_X+2][BLOCK_Y+2];
//    // clear out shared memory (zero padding)
//    if (is_x_top)           SMEM(tx-1, ty  ) = 0;
//    else if (is_x_bot)      SMEM(tx+1, ty  ) = 0;
//    if (is_y_top) {         SMEM(tx  , ty-1) = 0;
//        if (is_x_top)       SMEM(tx-1, ty-1) = 0;
//        else if (is_x_bot)  SMEM(tx+1, ty-1) = 0;
//    } else if (is_y_bot) {  SMEM(tx  , ty+1) = 0;
//        if (is_x_top)       SMEM(tx-1, ty+1) = 0;
//        else if (is_x_bot)  SMEM(tx+1, ty+1) = 0;
//    }
//
//    // guards: is at boundary and still more image?
//    int x = blockIdx.x * blockDim.x + tx;
//    int y = blockIdx.y * blockDim.y + ty;
//    is_x_top &= (x > 0); is_x_bot &= (x < nx - 1);
//    is_y_top &= (y > 0); is_y_bot &= (y < ny - 1);
//
//    // each thread pulls from image
//                            SMEM(tx  , ty  ) = IN(x  , y  ); // self
//    if (is_x_top)           SMEM(tx-1, ty  ) = IN(x-1, y  );
//    else if (is_x_bot)      SMEM(tx+1, ty  ) = IN(x+1, y  );
//    if (is_y_top) {         SMEM(tx  , ty-1) = IN(x  , y-1);
//        if (is_x_top)       SMEM(tx-1, ty-1) = IN(x-1, y-1);
//        else if (is_x_bot)  SMEM(tx+1, ty-1) = IN(x+1, y-1);
//    } else if (is_y_bot) {  SMEM(tx  , ty+1) = IN(x  , y+1);
//        if (is_x_top)       SMEM(tx-1, ty+1) = IN(x-1, y+1);
//        else if (is_x_bot)  SMEM(tx+1, ty+1) = IN(x+1, y+1);
//    }
//    __syncthreads();
//
//    // pull top six from shared memory
//    float v[6] = { SMEM(tx-1, ty-1), SMEM(tx  , ty-1), SMEM(tx+1, ty-1),
//                   SMEM(tx-1, ty  ), SMEM(tx  , ty  ), SMEM(tx+1, ty  ) };
//
//    // with each pass, remove min and max values and add new value
//    mnmx6(v[0], v[1], v[2], v[3], v[4], v[5]);
//    v[5] = SMEM(tx-1, ty+1); // add new contestant
//    mnmx5(v[1], v[2], v[3], v[4], v[5]);
//    v[5] = SMEM(tx  , ty+1);
//    mnmx4(v[2], v[3], v[4], v[5]);
//    v[5] = SMEM(tx+1, ty+1);
//    mnmx3(v[3], v[4], v[5]);
//
////    printf("the x is %d, y is %d, result is %f \n", x, y, v[4] );
//
//    // pick the middle one
//    d_out[y*nx + x] = v[4];
//}
