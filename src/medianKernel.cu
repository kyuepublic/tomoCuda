#include "hip/hip_runtime.h"
// Only support reflect mode right now


#include <medianFilter.hh>

//#define SMEM(X,Y)  smem[(X)+7][(Y)+7]
#define IN(X,Y)  d_in[(X)+(Y)*(14+nx)]

#define INO(X,Y) d_in[(X)+(Y)*(1+nx)]
#define INT(X,Y) d_in[(X)+(Y)*(2+nx)]
#define INTH(X,Y) d_in[(X)+(Y)*(3+nx)]
#define INF(X,Y) d_in[(X)+(Y)*(4+nx)]
#define INFI(X,Y) d_in[(X)+(Y)*(5+nx)]
#define INS(X,Y) d_in[(X)+(Y)*(6+nx)]
#define INSE(X,Y) d_in[(X)+(Y)*(7+nx)] // 8
#define INEI(X,Y) d_in[(X)+(Y)*(8+nx)] // 9

#define INEL(X,Y) d_in[(X)+(Y)*(10+nx)] // 11

#define INTHI(X,Y) d_in[(X)+(Y)*(12+nx)] // 13


#define INZ(X,Y,Z)  d_in[(X)+(Y)*(14+nx)+(Z)*(14+nx)*ny]

#define swapd(a,b)    { float tmp = a; a = min(a,b); b = max(tmp,b); }

#define SMEM(x,y)  smem[(x)+1][(y)+1]

__global__ void kernel2ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 2;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[4]={INO(x-1, y-1), INO(x-1, y),
                    INO(x, y-1), INO(x,y)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INO(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

//        for(int k = 1; k <winSize/2; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < winSize/2; i++) {
//                swapd(v[i], v[winSize-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= winSize/2; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = winSize-k-2; i >= winSize/2; i--) {
//                swapd(v[i], v[winSize-1-k]);
//            }
//        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[2];
    }

}

__global__ void kernel3ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 3;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[6]={INT(x-1, y-1), INT(x-1, y), INT(x-1, y+1),
                      INT(x, y-1), INT(x, y), INT(x, y+1)};


        const int ARR_SIZE = 6;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 1; k++) {

            for(int j = -1; j <= 1; j++) {

                // add new contestant to first position in array
                v[0] = INT(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k < 1; k++) {
            // move max/min into respective halves
            for(int i = k; i < 1; i++) {
                swapd(v[i], v[3-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 1; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 3-k-2; i >= 1; i--) {
                swapd(v[i], v[3-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[1];
    }

}

__global__ void kernel4ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 4;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[12]={INTH(x-2,y-2), INTH(x-2, y-1), INTH(x-2, y), INTH(x-2,y+1),
                    INTH(x-1,y-2), INTH(x-1, y-1), INTH(x-1, y), INTH(x-1,y+1),
                    INTH(x, y-2), INTH(x, y-1), INTH(x,y), INTH(x,y+1)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INTH(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize];
    }

}

__global__ void kernel5ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 5;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[15]={  INF(x-2, y-2), INF(x-2, y-1), INF(x-2, y), INF(x-2, y+1), INF(x-2, y+2),
                      INF(x-1, y-2), INF(x-1, y-1), INF(x-1, y), INF(x-1, y+1), INF(x-1, y+2),
                      INF(x, y-2), INF(x, y-1), INF(x, y), INF(x, y+1), INF(x, y+2)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INF(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}

__global__ void kernel6ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 6;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[24]={INFI(x-3,y-3),INFI(x-3,y-2), INFI(x-3, y-1), INFI(x-3, y), INFI(x-3,y+1), INFI(x-3,y+2),
                    INFI(x-2,y-3),INFI(x-2,y-2), INFI(x-2, y-1), INFI(x-2, y), INFI(x-2,y+1), INFI(x-2,y+2),
                    INFI(x-1,y-3),INFI(x-1,y-2), INFI(x-1, y-1), INFI(x-1, y), INFI(x-1,y+1),INFI(x-1,y+2),
                    INFI(x, y-3), INFI(x, y-2), INFI(x, y-1), INFI(x,y), INFI(x,y+1), INFI(x,y+2)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INFI(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize];
    }

}


__global__ void kernel7ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 7;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[28]={ INS(x-3, y-3), INS(x-3, y-2), INS(x-3, y-1), INS(x-3, y), INS(x-3, y+1), INS(x-3, y+2), INS(x-3, y+3),
                      INS(x-2, y-3), INS(x-2, y-2), INS(x-2, y-1), INS(x-2, y), INS(x-2, y+1), INS(x-2, y+2), INS(x-2, y+3),
                      INS(x-1, y-3), INS(x-1, y-2), INS(x-1, y-1), INS(x-1, y), INS(x-1, y+1), INS(x-1, y+2), INS(x-1, y+3),
                      INS(x, y-3), INS(x, y-2), INS(x, y-1), INS(x, y), INS(x, y+1), INS(x, y+2), INS(x, y+3)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INS(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}

__global__ void kernel8ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 8;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[40]={INSE(x-4,y-4),INSE(x-4,y-3),INSE(x-4,y-2), INSE(x-4, y-1), INSE(x-4, y), INSE(x-4,y+1), INSE(x-4,y+2),INSE(x-4,y+3),
                    INSE(x-3,y-4),INSE(x-3,y-3),INSE(x-3,y-2), INSE(x-3, y-1), INSE(x-3, y), INSE(x-3,y+1), INSE(x-3,y+2),INSE(x-3,y+3),
                    INSE(x-2,y-4),INSE(x-2,y-3),INSE(x-2,y-2), INSE(x-2, y-1), INSE(x-2, y), INSE(x-2,y+1), INSE(x-2,y+2),INSE(x-2,y+3),
                    INSE(x-1,y-4),INSE(x-1,y-3),INSE(x-1,y-2), INSE(x-1, y-1), INSE(x-1, y), INSE(x-1,y+1),INSE(x-1,y+2),INSE(x-1,y+3),
                    INSE(x, y-4),INSE(x, y-3), INSE(x, y-2), INSE(x, y-1), INSE(x,y), INSE(x,y+1), INSE(x,y+2), INSE(x,y+3)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INSE(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize];
    }

}

__global__ void kernel9ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 9;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[45]={ INEI(x-4, y-4),INEI(x-4, y-3), INEI(x-4, y-2), INEI(x-4, y-1), INEI(x-4, y), INEI(x-4, y+1), INEI(x-4, y+2), INEI(x-4, y+3),INEI(x-4, y+4),
                      INEI(x-3, y-4),INEI(x-3, y-3), INEI(x-3, y-2), INEI(x-3, y-1), INEI(x-3, y), INEI(x-3, y+1), INEI(x-3, y+2), INEI(x-3, y+3),INEI(x-3, y+4),
                      INEI(x-2, y-4),INEI(x-2, y-3), INEI(x-2, y-2), INEI(x-2, y-1), INEI(x-2, y), INEI(x-2, y+1), INEI(x-2, y+2), INEI(x-2, y+3),INEI(x-2, y+4),
                      INEI(x-1, y-4),INEI(x-1, y-3), INEI(x-1, y-2), INEI(x-1, y-1), INEI(x-1, y), INEI(x-1, y+1), INEI(x-1, y+2), INEI(x-1, y+3),INEI(x-1, y+4),
                      INEI(x, y-4),INEI(x, y-3), INEI(x, y-2), INEI(x, y-1), INEI(x, y), INEI(x, y+1), INEI(x, y+2), INEI(x, y+3),INEI(x, y+4)};



        const int ARR_SIZE = winSize*(winSize-winSize/2); // float array size

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INEI(x+k, y+j); // change window size change here

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}

__global__ void kernel11ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 11;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[66]={ INEL(x-5, y-5),INEL(x-5, y-4),INEL(x-5, y-3), INEL(x-5, y-2), INEL(x-5, y-1), INEL(x-5, y), INEL(x-5, y+1), INEL(x-5, y+2), INEL(x-5, y+3),INEL(x-5, y+4),INEL(x-5, y+5),
                      INEL(x-4, y-5),INEL(x-4, y-4),INEL(x-4, y-3), INEL(x-4, y-2), INEL(x-4, y-1), INEL(x-4, y), INEL(x-4, y+1), INEL(x-4, y+2), INEL(x-4, y+3),INEL(x-4, y+4),INEL(x-4, y+5),
                      INEL(x-3, y-5),INEL(x-3, y-4),INEL(x-3, y-3), INEL(x-3, y-2), INEL(x-3, y-1), INEL(x-3, y), INEL(x-3, y+1), INEL(x-3, y+2), INEL(x-3, y+3),INEL(x-3, y+4),INEL(x-3, y+5),
                      INEL(x-2, y-5),INEL(x-2, y-4),INEL(x-2, y-3), INEL(x-2, y-2), INEL(x-2, y-1), INEL(x-2, y), INEL(x-2, y+1), INEL(x-2, y+2), INEL(x-2, y+3),INEL(x-2, y+4),INEL(x-2, y+5),
                      INEL(x-1, y-5),INEL(x-1, y-4),INEL(x-1, y-3), INEL(x-1, y-2), INEL(x-1, y-1), INEL(x-1, y), INEL(x-1, y+1), INEL(x-1, y+2), INEL(x-1, y+3),INEL(x-1, y+4),INEL(x-1, y+5),
                      INEL(x, y-5),INEL(x, y-4),INEL(x, y-3), INEL(x, y-2), INEL(x, y-1), INEL(x, y), INEL(x, y+1), INEL(x, y+2), INEL(x, y+3),INEL(x, y+4),INEL(x, y+5)};



        const int ARR_SIZE = winSize*(winSize-winSize/2); // float array size

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INEL(x+k, y+j); // change window size change here

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}


__global__ void kernel13ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 13;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[91]={ INTHI(x-6, y-6),INTHI(x-6, y-5),INTHI(x-6, y-4),INTHI(x-6, y-3), INTHI(x-6, y-2), INTHI(x-6, y-1), INTHI(x-6, y), INTHI(x-6, y+1), INTHI(x-6, y+2), INTHI(x-6, y+3),INTHI(x-6, y+4),INTHI(x-6, y+5),INTHI(x-6, y+6),
                      INTHI(x-5, y-6),INTHI(x-5, y-5),INTHI(x-5, y-4),INTHI(x-5, y-3), INTHI(x-5, y-2), INTHI(x-5, y-1), INTHI(x-5, y), INTHI(x-5, y+1), INTHI(x-5, y+2), INTHI(x-5, y+3),INTHI(x-5, y+4),INTHI(x-5, y+5),INTHI(x-5, y+6),
                      INTHI(x-4, y-6),INTHI(x-4, y-5),INTHI(x-4, y-4),INTHI(x-4, y-3), INTHI(x-4, y-2), INTHI(x-4, y-1), INTHI(x-4, y), INTHI(x-4, y+1), INTHI(x-4, y+2), INTHI(x-4, y+3),INTHI(x-4, y+4),INTHI(x-4, y+5),INTHI(x-4, y+6),
                      INTHI(x-3, y-6),INTHI(x-3, y-5),INTHI(x-3, y-4),INTHI(x-3, y-3), INTHI(x-3, y-2), INTHI(x-3, y-1), INTHI(x-3, y), INTHI(x-3, y+1), INTHI(x-3, y+2), INTHI(x-3, y+3),INTHI(x-3, y+4),INTHI(x-3, y+5),INTHI(x-3, y+6),
                      INTHI(x-2, y-6),INTHI(x-2, y-5),INTHI(x-2, y-4),INTHI(x-2, y-3), INTHI(x-2, y-2), INTHI(x-2, y-1), INTHI(x-2, y), INTHI(x-2, y+1), INTHI(x-2, y+2), INTHI(x-2, y+3),INTHI(x-2, y+4),INTHI(x-2, y+5),INTHI(x-2, y+6),
                      INTHI(x-1, y-6),INTHI(x-1, y-5),INTHI(x-1, y-4),INTHI(x-1, y-3), INTHI(x-1, y-2), INTHI(x-1, y-1), INTHI(x-1, y), INTHI(x-1, y+1), INTHI(x-1, y+2), INTHI(x-1, y+3),INTHI(x-1, y+4),INTHI(x-1, y+5),INTHI(x-1, y+6),
                      INTHI(x, y-6),INTHI(x, y-5),INTHI(x, y-4),INTHI(x, y-3), INTHI(x, y-2), INTHI(x, y-1), INTHI(x, y), INTHI(x, y+1), INTHI(x, y+2), INTHI(x, y+3),INTHI(x, y+4),INTHI(x, y+5),INTHI(x, y+6)};



        const int ARR_SIZE = winSize*(winSize-winSize/2); // float array size

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INTHI(x+k, y+j); // change window size change here

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}

// Use the new exchange way

__global__ void kernel15ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[120]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7)
                    };


        const int ARR_SIZE = 120;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 7; k++) {

            for(int j = -7; j <= 7; j++) {

                // add new contestant to first position in array
                v[0] = IN(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[7];
    }

}

__global__ void reomveOutliner2D2ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 2;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[4]={INO(x-1, y-1), INO(x-1, y),
                    INO(x, y-1), INO(x,y)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INO(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

//        for(int k = 1; k <winSize/2; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < winSize/2; i++) {
//                swapd(v[i], v[winSize-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= winSize/2; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = winSize-k-2; i >= winSize/2; i--) {
//                swapd(v[i], v[winSize-1-k]);
//            }
//        }

        float currentPixel = INO(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D3ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 3;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[6]={INT(x-1, y-1), INT(x-1, y), INT(x-1, y+1),
                      INT(x, y-1), INT(x, y), INT(x, y+1)};


        const int ARR_SIZE = 6;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 1; k++) {

            for(int j = -1; j <= 1; j++) {

                // add new contestant to first position in array
                v[0] = INT(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        float currentPixel = INT(x, y);

        int mask = 0;
        if((currentPixel-v[1]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[1]*mask+currentPixel*(1-mask);
    }

}

__global__ void reomveOutliner2D4ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 4;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[12]={INTH(x-2,y-2), INTH(x-2, y-1), INTH(x-2, y), INTH(x-2,y+1),
                    INTH(x-1,y-2), INTH(x-1, y-1), INTH(x-1, y), INTH(x-1,y+1),
                    INTH(x, y-2), INTH(x, y-1), INTH(x,y), INTH(x,y+1)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INTH(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        float currentPixel = INTH(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D5ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 5;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[15]={  INF(x-2, y-2), INF(x-2, y-1), INF(x-2, y), INF(x-2, y+1), INF(x-2, y+2),
                      INF(x-1, y-2), INF(x-1, y-1), INF(x-1, y), INF(x-1, y+1), INF(x-1, y+2),
                      INF(x, y-2), INF(x, y-1), INF(x, y), INF(x, y+1), INF(x, y+2)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INF(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        float currentPixel = INF(x, y);

        int mask = 0;
        if((currentPixel-v[2]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[2]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D6ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 6;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[24]={INFI(x-3,y-3),INFI(x-3,y-2), INFI(x-3, y-1), INFI(x-3, y), INFI(x-3,y+1), INFI(x-3,y+2),
                    INFI(x-2,y-3),INFI(x-2,y-2), INFI(x-2, y-1), INFI(x-2, y), INFI(x-2,y+1), INFI(x-2,y+2),
                    INFI(x-1,y-3),INFI(x-1,y-2), INFI(x-1, y-1), INFI(x-1, y), INFI(x-1,y+1),INFI(x-1,y+2),
                    INFI(x, y-3), INFI(x, y-2), INFI(x, y-1), INFI(x,y), INFI(x,y+1), INFI(x,y+2)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INFI(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        float currentPixel = INFI(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);
    }

}



__global__ void reomveOutliner2D7ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 7;

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[28]={ INS(x-3, y-3), INS(x-3, y-2), INS(x-3, y-1), INS(x-3, y), INS(x-3, y+1), INS(x-3, y+2), INS(x-3, y+3),
                      INS(x-2, y-3), INS(x-2, y-2), INS(x-2, y-1), INS(x-2, y), INS(x-2, y+1), INS(x-2, y+2), INS(x-2, y+3),
                      INS(x-1, y-3), INS(x-1, y-2), INS(x-1, y-1), INS(x-1, y), INS(x-1, y+1), INS(x-1, y+2), INS(x-1, y+3),
                      INS(x, y-3), INS(x, y-2), INS(x, y-1), INS(x, y), INS(x, y+1), INS(x, y+2), INS(x, y+3)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INS(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        float currentPixel = INS(x, y);

        int mask = 0;
        if((currentPixel-v[3]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[3]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D8ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 8;
//        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[40]={INSE(x-4,y-4),INSE(x-4,y-3),INSE(x-4,y-2), INSE(x-4, y-1), INSE(x-4, y), INSE(x-4,y+1), INSE(x-4,y+2),INSE(x-4,y+3),
                    INSE(x-3,y-4),INSE(x-3,y-3),INSE(x-3,y-2), INSE(x-3, y-1), INSE(x-3, y), INSE(x-3,y+1), INSE(x-3,y+2),INSE(x-3,y+3),
                    INSE(x-2,y-4),INSE(x-2,y-3),INSE(x-2,y-2), INSE(x-2, y-1), INSE(x-2, y), INSE(x-2,y+1), INSE(x-2,y+2),INSE(x-2,y+3),
                    INSE(x-1,y-4),INSE(x-1,y-3),INSE(x-1,y-2), INSE(x-1, y-1), INSE(x-1, y), INSE(x-1,y+1),INSE(x-1,y+2),INSE(x-1,y+3),
                    INSE(x, y-4),INSE(x, y-3), INSE(x, y-2), INSE(x, y-1), INSE(x,y), INSE(x,y+1), INSE(x,y+2), INSE(x,y+3)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INSE(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }


        float currentPixel = INSE(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);

    }

}

__global__ void reomveOutliner2D15ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
//        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[120]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7)
                    };


        const int ARR_SIZE = 120;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 7; k++) {

            for(int j = -7; j <= 7; j++) {

                // add new contestant to first position in array
                v[0] = IN(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[7];

        float currentPixel = IN(x, y);

        int mask = 0;
        if((currentPixel-v[7]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[7]*mask+currentPixel*(1-mask);

    }

}





