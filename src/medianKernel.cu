#include "hip/hip_runtime.h"
// Only support reflect mode right now


#include <medianFilter.hh>

//#define SMEM(X,Y)  smem[(X)+7][(Y)+7]
#define IN(X,Y)  d_in[(X)+(Y)*(14+nx)]

#define INO(X,Y) d_in[(X)+(Y)*(1+nx)]
#define INT(X,Y) d_in[(X)+(Y)*(2+nx)]
#define INTH(X,Y) d_in[(X)+(Y)*(3+nx)]
#define INF(X,Y) d_in[(X)+(Y)*(4+nx)]
#define INFI(X,Y) d_in[(X)+(Y)*(5+nx)]
#define INS(X,Y) d_in[(X)+(Y)*(6+nx)]
#define INSE(X,Y) d_in[(X)+(Y)*(7+nx)]

#define INZ(X,Y,Z)  d_in[(X)+(Y)*(14+nx)+(Z)*(14+nx)*ny]

#define swapd(a,b)    { float tmp = a; a = min(a,b); b = max(tmp,b); }

#define SMEM(x,y)  smem[(x)+1][(y)+1]

__global__ void kernel2ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 2;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[4]={INO(x-1, y-1), INO(x-1, y),
                    INO(x, y-1), INO(x,y)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INO(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

//        for(int k = 1; k <winSize/2; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < winSize/2; i++) {
//                swapd(v[i], v[winSize-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= winSize/2; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = winSize-k-2; i >= winSize/2; i--) {
//                swapd(v[i], v[winSize-1-k]);
//            }
//        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[2];
    }

}

__global__ void kernel3ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 3;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[6]={INT(x-1, y-1), INT(x-1, y), INT(x-1, y+1),
                      INT(x, y-1), INT(x, y), INT(x, y+1)};


        const int ARR_SIZE = 6;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 1; k++) {

            for(int j = -1; j <= 1; j++) {

                // add new contestant to first position in array
                v[0] = INT(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k < 1; k++) {
            // move max/min into respective halves
            for(int i = k; i < 1; i++) {
                swapd(v[i], v[3-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 1; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 3-k-2; i >= 1; i--) {
                swapd(v[i], v[3-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[1];
    }

}

__global__ void kernel4ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 4;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[12]={INTH(x-2,y-2), INTH(x-2, y-1), INTH(x-2, y), INTH(x-2,y+1),
                    INTH(x-1,y-2), INTH(x-1, y-1), INTH(x-1, y), INTH(x-1,y+1),
                    INTH(x, y-2), INTH(x, y-1), INTH(x,y), INTH(x,y+1)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INTH(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize];
    }

}

__global__ void kernel5ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 5;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[15]={  INF(x-2, y-2), INF(x-2, y-1), INF(x-2, y), INF(x-2, y+1), INF(x-2, y+2),
                      INF(x-1, y-2), INF(x-1, y-1), INF(x-1, y), INF(x-1, y+1), INF(x-1, y+2),
                      INF(x, y-2), INF(x, y-1), INF(x, y), INF(x, y+1), INF(x, y+2)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INF(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}

__global__ void kernel6ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 6;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[24]={INFI(x-3,y-3),INFI(x-3,y-2), INFI(x-3, y-1), INFI(x-3, y), INFI(x-3,y+1), INFI(x-3,y+2),
                    INFI(x-2,y-3),INFI(x-2,y-2), INFI(x-2, y-1), INFI(x-2, y), INFI(x-2,y+1), INFI(x-2,y+2),
                    INFI(x-1,y-3),INFI(x-1,y-2), INFI(x-1, y-1), INFI(x-1, y), INFI(x-1,y+1),INFI(x-1,y+2),
                    INFI(x, y-3), INFI(x, y-2), INFI(x, y-1), INFI(x,y), INFI(x,y+1), INFI(x,y+2)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INFI(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize];
    }

}


__global__ void kernel7ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 7;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[28]={ INS(x-3, y-3), INS(x-3, y-2), INS(x-3, y-1), INS(x-3, y), INS(x-3, y+1), INS(x-3, y+2), INS(x-3, y+3),
                      INS(x-2, y-3), INS(x-2, y-2), INS(x-2, y-1), INS(x-2, y), INS(x-2, y+1), INS(x-2, y+2), INS(x-2, y+3),
                      INS(x-1, y-3), INS(x-1, y-2), INS(x-1, y-1), INS(x-1, y), INS(x-1, y+1), INS(x-1, y+2), INS(x-1, y+3),
                      INS(x, y-3), INS(x, y-2), INS(x, y-1), INS(x, y), INS(x, y+1), INS(x, y+2), INS(x, y+3)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INS(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize/2];
    }

}

__global__ void kernel8ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 8;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[40]={INSE(x-4,y-4),INSE(x-4,y-3),INSE(x-4,y-2), INSE(x-4, y-1), INSE(x-4, y), INSE(x-4,y+1), INSE(x-4,y+2),INSE(x-4,y+3),
                    INSE(x-3,y-4),INSE(x-3,y-3),INSE(x-3,y-2), INSE(x-3, y-1), INSE(x-3, y), INSE(x-3,y+1), INSE(x-3,y+2),INSE(x-3,y+3),
                    INSE(x-2,y-4),INSE(x-2,y-3),INSE(x-2,y-2), INSE(x-2, y-1), INSE(x-2, y), INSE(x-2,y+1), INSE(x-2,y+2),INSE(x-2,y+3),
                    INSE(x-1,y-4),INSE(x-1,y-3),INSE(x-1,y-2), INSE(x-1, y-1), INSE(x-1, y), INSE(x-1,y+1),INSE(x-1,y+2),INSE(x-1,y+3),
                    INSE(x, y-4),INSE(x, y-3), INSE(x, y-2), INSE(x, y-1), INSE(x,y), INSE(x,y+1), INSE(x,y+2), INSE(x,y+3)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INSE(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize];
    }

}


// Use the new exchange way

__global__ void kernel15ME(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[120]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7)
                    };


        const int ARR_SIZE = 120;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 7; k++) {

            for(int j = -7; j <= 7; j++) {

                // add new contestant to first position in array
                v[0] = IN(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[7];
    }

}

__global__ void reomveOutliner2D2ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 2;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[4]={INO(x-1, y-1), INO(x-1, y),
                    INO(x, y-1), INO(x,y)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INO(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

//        for(int k = 1; k <winSize/2; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < winSize/2; i++) {
//                swapd(v[i], v[winSize-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= winSize/2; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = winSize-k-2; i >= winSize/2; i--) {
//                swapd(v[i], v[winSize-1-k]);
//            }
//        }

        float currentPixel = INO(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D3ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 3;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[6]={INT(x-1, y-1), INT(x-1, y), INT(x-1, y+1),
                      INT(x, y-1), INT(x, y), INT(x, y+1)};


        const int ARR_SIZE = 6;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 1; k++) {

            for(int j = -1; j <= 1; j++) {

                // add new contestant to first position in array
                v[0] = INT(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        float currentPixel = INT(x, y);

        int mask = 0;
        if((currentPixel-v[1]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[1]*mask+currentPixel*(1-mask);
    }

}

__global__ void reomveOutliner2D4ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 4;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[12]={INTH(x-2,y-2), INTH(x-2, y-1), INTH(x-2, y), INTH(x-2,y+1),
                    INTH(x-1,y-2), INTH(x-1, y-1), INTH(x-1, y), INTH(x-1,y+1),
                    INTH(x, y-2), INTH(x, y-1), INTH(x,y), INTH(x,y+1)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INTH(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        float currentPixel = INTH(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D5ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 5;

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[15]={  INF(x-2, y-2), INF(x-2, y-1), INF(x-2, y), INF(x-2, y+1), INF(x-2, y+2),
                      INF(x-1, y-2), INF(x-1, y-1), INF(x-1, y), INF(x-1, y+1), INF(x-1, y+2),
                      INF(x, y-2), INF(x, y-1), INF(x, y), INF(x, y+1), INF(x, y+2)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INF(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        float currentPixel = INF(x, y);

        int mask = 0;
        if((currentPixel-v[2]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[2]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D6ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 6;
//        float v[225] = {0};

//        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[24]={INFI(x-3,y-3),INFI(x-3,y-2), INFI(x-3, y-1), INFI(x-3, y), INFI(x-3,y+1), INFI(x-3,y+2),
                    INFI(x-2,y-3),INFI(x-2,y-2), INFI(x-2, y-1), INFI(x-2, y), INFI(x-2,y+1), INFI(x-2,y+2),
                    INFI(x-1,y-3),INFI(x-1,y-2), INFI(x-1, y-1), INFI(x-1, y), INFI(x-1,y+1),INFI(x-1,y+2),
                    INFI(x, y-3), INFI(x, y-2), INFI(x, y-1), INFI(x,y), INFI(x,y+1), INFI(x,y+2)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INFI(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }

        float currentPixel = INFI(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);
    }

}



__global__ void reomveOutliner2D7ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 7;

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[28]={ INS(x-3, y-3), INS(x-3, y-2), INS(x-3, y-1), INS(x-3, y), INS(x-3, y+1), INS(x-3, y+2), INS(x-3, y+3),
                      INS(x-2, y-3), INS(x-2, y-2), INS(x-2, y-1), INS(x-2, y), INS(x-2, y+1), INS(x-2, y+2), INS(x-2, y+3),
                      INS(x-1, y-3), INS(x-1, y-2), INS(x-1, y-1), INS(x-1, y), INS(x-1, y+1), INS(x-1, y+2), INS(x-1, y+3),
                      INS(x, y-3), INS(x, y-2), INS(x, y-1), INS(x, y), INS(x, y+1), INS(x, y+2), INS(x, y+3)};


        const int ARR_SIZE = winSize*(winSize-winSize/2);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2; k++) {

            for(int j = -winSize/2; j <= winSize/2; j++) {

                // add new contestant to first position in array
                v[0] = INS(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k <winSize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < winSize/2; i++) {
                swapd(v[i], v[winSize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= winSize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = winSize-k-2; i >= winSize/2; i--) {
                swapd(v[i], v[winSize-1-k]);
            }
        }

        float currentPixel = INS(x, y);

        int mask = 0;
        if((currentPixel-v[3]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[3]*mask+currentPixel*(1-mask);

    }

}


__global__ void reomveOutliner2D8ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 8;
//        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[40]={INSE(x-4,y-4),INSE(x-4,y-3),INSE(x-4,y-2), INSE(x-4, y-1), INSE(x-4, y), INSE(x-4,y+1), INSE(x-4,y+2),INSE(x-4,y+3),
                    INSE(x-3,y-4),INSE(x-3,y-3),INSE(x-3,y-2), INSE(x-3, y-1), INSE(x-3, y), INSE(x-3,y+1), INSE(x-3,y+2),INSE(x-3,y+3),
                    INSE(x-2,y-4),INSE(x-2,y-3),INSE(x-2,y-2), INSE(x-2, y-1), INSE(x-2, y), INSE(x-2,y+1), INSE(x-2,y+2),INSE(x-2,y+3),
                    INSE(x-1,y-4),INSE(x-1,y-3),INSE(x-1,y-2), INSE(x-1, y-1), INSE(x-1, y), INSE(x-1,y+1),INSE(x-1,y+2),INSE(x-1,y+3),
                    INSE(x, y-4),INSE(x, y-3), INSE(x, y-2), INSE(x, y-1), INSE(x,y), INSE(x,y+1), INSE(x,y+2), INSE(x,y+3)};


        const int ARR_SIZE = winSize*(winSize/2+1);

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= winSize/2-1; k++) {

            for(int j = -winSize/2; j <= winSize/2-1; j++) {

                // add new contestant to first position in array
                v[0] = INSE(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        int fvecsize = 2*winSize;

        for(int k = 1; k <fvecsize/2; k++) {
            // move max/min into respective halves
            for(int i = k; i < fvecsize/2; i++) {
                swapd(v[i], v[fvecsize-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= fvecsize/2; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = fvecsize-k-2; i >= fvecsize/2; i--) {
                swapd(v[i], v[fvecsize-1-k]);
            }
        }


        float currentPixel = INSE(x, y);

        int mask = 0;
        if((currentPixel-v[winSize]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[winSize]*mask+currentPixel*(1-mask);

    }

}

__global__ void reomveOutliner2D15ME(int nx, int ny, int diff, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < nx) && (y < ny))
    {
        int winSize = 15;
//        float v[225] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = (winSize-1)/2;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;

// use macro to fetch the value, like loop unrolling
        float v[120]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7)
                    };


        const int ARR_SIZE = 120;

#pragma unroll
        for(int i = 0; i < ARR_SIZE/2; i++) {
            swapd(v[i], v[ARR_SIZE-1-i]);
        }

#pragma unroll
        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
            swapd(v[0], v[i]);
        }

#pragma unroll
        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
            swapd(v[i], v[ARR_SIZE-1]);
        }

        int last = ARR_SIZE-1;

        for(int k = 1; k <= 7; k++) {

            for(int j = -7; j <= 7; j++) {

                // add new contestant to first position in array
                v[0] = IN(x+k, y+j);

                last--;

                // place max in last half, min in first half
                for(int i = 0; i < (last+1)/2; i++) {
                    swapd(v[i], v[last-i]);
                }
                // now perform swaps on each half such that
                // max is in last pos, min is in first pos
                for(int i = 1; i <= last/2; i++) {
                    swapd(v[0], v[i]);
                }
                for(int i = last-1; i >= (last+1)/2; i--) {
                    swapd(v[i], v[last]);
                }
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        for(int k = 1; k < 7; k++) {
            // move max/min into respective halves
            for(int i = k; i < 7; i++) {
                swapd(v[i], v[15-1-i]);
            }
            // move min into first pos
            for(int i = k+1; i <= 7; i++) {
                swapd(v[k], v[i]);
            }
            // move max into last pos
            for(int i = 15-k-2; i >= 7; i--) {
                swapd(v[i], v[15-1-k]);
            }
        }

        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[7];

        float currentPixel = IN(x, y);

        int mask = 0;
        if((currentPixel-v[7]) >= diff)
            mask = 1;
        else
            mask = 0;

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[7]*mask+currentPixel*(1-mask);

    }

}

//__global__ void kernel3MES(int nx, int ny, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    int tx = threadIdx.x, ty = threadIdx.y;
//
//    bool is_x_top = (tx == 0), is_x_bot = (tx == BLOCK_X-1);
//    bool is_y_top = (ty == 0), is_y_bot = (ty == BLOCK_Y-1);
//
//
//    __shared__ float smem[BLOCK_X+2][BLOCK_Y+2];
//
//    // clear out shared memory (zero padding)
//    if (is_x_top)           SMEM(tx-1, ty  ) = 0;
//    else if (is_x_bot)      SMEM(tx+1, ty  ) = 0;
//    if (is_y_top) {         SMEM(tx  , ty-1) = 0;
//        if (is_x_top)       SMEM(tx-1, ty-1) = 0;
//        else if (is_x_bot)  SMEM(tx+1, ty-1) = 0;
//    } else if (is_y_bot) {  SMEM(tx  , ty+1) = 0;
//        if (is_x_top)       SMEM(tx-1, ty+1) = 0;
//        else if (is_x_bot)  SMEM(tx+1, ty+1) = 0;
//    }
//
//
//    is_x_top &= (x > 0); is_x_bot &= (x < nx - 1);
//    is_y_top &= (y > 0); is_y_bot &= (y < ny - 1);
//
//    int winSize = 3;
//
//    int vecSize = winSize*winSize;
//    int loffset = winSize/2;
//    int roffset = (winSize-1)/2;
//    int toffset = loffset+roffset;
//
//    x = x + loffset;
//    y = y + loffset;
//
//
//                            SMEM(tx  , ty  ) = INT(x  , y  ); // self
//    if (is_x_top)           SMEM(tx-1, ty  ) = INT(x-1, y  );
//    else if (is_x_bot)      SMEM(tx+1, ty  ) = INT(x+1, y  );
//    if (is_y_top) {         SMEM(tx  , ty-1) = INT(x  , y-1);
//        if (is_x_top)       SMEM(tx-1, ty-1) = INT(x-1, y-1);
//        else if (is_x_bot)  SMEM(tx+1, ty-1) = INT(x+1, y-1);
//    } else if (is_y_bot) {  SMEM(tx  , ty+1) = INT(x  , y+1);
//        if (is_x_top)       SMEM(tx-1, ty+1) = INT(x-1, y+1);
//        else if (is_x_bot)  SMEM(tx+1, ty+1) = INT(x+1, y+1);
//    }
//    __syncthreads();
//
//    if ((x < nx) && (y < ny))
//    {
//
//// use macro to fetch the value, like loop unrolling
//        float v[6] = { SMEM(tx-1, ty-1), SMEM(tx  , ty-1), SMEM(tx+1, ty-1),
//                   SMEM(tx-1, ty  ), SMEM(tx  , ty  ), SMEM(tx+1, ty  ) };
//
//
//        const int ARR_SIZE = 6;
//
//#pragma unroll
//        for(int i = 0; i < ARR_SIZE/2; i++) {
//            swapd(v[i], v[ARR_SIZE-1-i]);
//        }
//
//#pragma unroll
//        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
//            swapd(v[0], v[i]);
//        }
//
//#pragma unroll
//        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
//            swapd(v[i], v[ARR_SIZE-1]);
//        }
//
//        int last = ARR_SIZE-1;
//
//        for(int k = 1; k <= 1; k++) {
//
//            for(int j = -1; j <= 1; j++) {
//
//                // add new contestant to first position in array
//                v[0] = SMEM(tx+j, ty+k);
//
//                last--;
//
//                // place max in last half, min in first half
//                for(int i = 0; i < (last+1)/2; i++) {
//                    swapd(v[i], v[last-i]);
//                }
//                // now perform swaps on each half such that
//                // max is in last pos, min is in first pos
//                for(int i = 1; i <= last/2; i++) {
//                    swapd(v[0], v[i]);
//                }
//                for(int i = last-1; i >= (last+1)/2; i--) {
//                    swapd(v[i], v[last]);
//                }
//            }
//        }
//
//        for(int k = 1; k < 1; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < 1; i++) {
//                swapd(v[i], v[3-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= 1; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = 3-k-2; i >= 1; i--) {
//                swapd(v[i], v[3-1-k]);
//            }
//        }
//
//        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[1];
//    }
//
//}

// windows size 4 byb 4
//__global__ void kernel4(int nx, int ny, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    int winSize = 4;
//    int loffset = winSize/2;
//    int roffset = winSize/2 - 1;
//    int toffset = loffset+roffset;
//
//    x = x + loffset;
//    y = y + loffset;
//
//    int i = 0;
//    float v[16] = {0};
//
//    for (int xx = x - loffset; xx <= x + roffset; xx++)
//    {
//        for (int yy = y - loffset; yy <= y + roffset; yy++)
//        {
//            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                v[i++] = d_in[yy*(nx+toffset) + xx];
//        }
//    }
//
//    // bubble-sort
//    for (int i = 0; i < 16; i++)
//    {
//        for (int j = i + 1; j < 16; j++)
//        {
//            if (v[i] > v[j])
//            { /* swap? */
//                float tmp = v[i];
//                v[i] = v[j];
//                v[j] = tmp;
//            }
//        }
//    }
//
//    // pick the middle one
//    d_out[(y-loffset)*nx + x-loffset] = v[8];
//}
//
//// Windows size 5 by b5
//__global__ void kernel5(int nx, int ny, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    int winSize = 5;
//    float v[25] = {0};
//
//    int vecSize = winSize*winSize;
//    int loffset = winSize/2;
//    int roffset = (winSize-1)/2;
//    int toffset = loffset+roffset;
//
//    x = x + loffset;
//    y = y + loffset;
//
//    int i = 0;
//
//    for (int xx = x - loffset; xx <= x + roffset; xx++)
//    {
//        for (int yy = y - loffset; yy <= y + roffset; yy++)
//        {
//            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                v[i++] = d_in[yy*(nx+toffset) + xx];
//        }
//    }
//
//    // bubble-sort
//    for (int i = 0; i < vecSize; i++)
//    {
//        for (int j = i + 1; j < vecSize; j++)
//        {
//            if (v[i] > v[j])
//            { /* swap? */
//                float tmp = v[i];
//                v[i] = v[j];
//                v[j] = tmp;
//            }
//        }
//    }
//
//    // pick the middle one
//    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
//
//}
//
//// windows size 6 byb 6
//__global__ void kernel6(int nx, int ny, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    int winSize = 6;
//    float v[36] = {0};
//
//    int vecSize = winSize*winSize;
//    int loffset = winSize/2;
//    int roffset = winSize/2 - 1;
//    int toffset = loffset+roffset;
//
//    x = x + loffset;
//    y = y + loffset;
//
//    int i = 0;
//
//
//    for (int xx = x - loffset; xx <= x + roffset; xx++)
//    {
//        for (int yy = y - loffset; yy <= y + roffset; yy++)
//        {
//            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                v[i++] = d_in[yy*(nx+toffset) + xx];
//        }
//    }
//
//    // bubble-sort
//    for (int i = 0; i < vecSize; i++)
//    {
//        for (int j = i + 1; j < vecSize; j++)
//        {
//            if (v[i] > v[j])
//            { /* swap? */
//                float tmp = v[i];
//                v[i] = v[j];
//                v[j] = tmp;
//            }
//        }
//    }
//
//    // pick the middle one
//    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
//}
//
//
//// window size 15 by b15
//__global__ void kernel15(int nx, int ny, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if ((x < nx) && (y < ny))
//    {
//        int winSize = 15;
//        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int yy = y - loffset; yy <= y + roffset; yy++)
//            {
////                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                    v[i++] = d_in[yy*(nx+toffset) + xx];
//            }
//        }
//
//        // bubble-sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                { /* swap? */
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
//    }
//
//}

//__global__ void kernel15M(int nx, int ny, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if ((x < nx) && (y < ny))
//    {
//        int winSize = 15;
////        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//
//        x = x + loffset;
//        y = y + loffset;
//
//// use macro to fetch the value, like loop unrolling
//        float v[225]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
//                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
//                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
//                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
//                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
//                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
//                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
//                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7),
//                    IN(x+1, y-7), IN(x+1, y-6), IN(x+1, y-5), IN(x+1, y-4), IN(x+1, y-3), IN(x+1, y-2), IN(x+1, y-1), IN(x+1, y), IN(x+1, y+1), IN(x+1, y+2), IN(x+1, y+3), IN(x+1, y+4), IN(x+1, y+5), IN(x+1, y+6), IN(x+1, y+7),
//                    IN(x+2, y-7), IN(x+2, y-6), IN(x+2, y-5), IN(x+2, y-4), IN(x+2, y-3), IN(x+2, y-2), IN(x+2, y-1), IN(x+2, y), IN(x+2, y+1), IN(x+2, y+2), IN(x+2, y+3), IN(x+2, y+4), IN(x+2, y+5), IN(x+2, y+6), IN(x+2, y+7),
//                    IN(x+3, y-7), IN(x+3, y-6), IN(x+3, y-5), IN(x+3, y-4), IN(x+3, y-3), IN(x+3, y-2), IN(x+3, y-1), IN(x+3, y), IN(x+3, y+1), IN(x+3, y+2), IN(x+3, y+3), IN(x+3, y+4), IN(x+3, y+5), IN(x+3, y+6), IN(x+3, y+7),
//                    IN(x+4, y-7), IN(x+4, y-6), IN(x+4, y-5), IN(x+4, y-4), IN(x+4, y-3), IN(x+4, y-2), IN(x+4, y-1), IN(x+4, y), IN(x+4, y+1), IN(x+4, y+2), IN(x+4, y+3), IN(x+4, y+4), IN(x+4, y+5), IN(x+4, y+6), IN(x+4, y+7),
//                    IN(x+5, y-7), IN(x+5, y-6), IN(x+5, y-5), IN(x+5, y-4), IN(x+5, y-3), IN(x+5, y-2), IN(x+5, y-1), IN(x+5, y), IN(x+5, y+1), IN(x+5, y+2), IN(x+5, y+3), IN(x+5, y+4), IN(x+5, y+5), IN(x+5, y+6), IN(x+5, y+7),
//                    IN(x+6, y-7), IN(x+6, y-6), IN(x+6, y-5), IN(x+6, y-4), IN(x+6, y-3), IN(x+6, y-2), IN(x+6, y-1), IN(x+6, y), IN(x+6, y+1), IN(x+6, y+2), IN(x+6, y+3), IN(x+6, y+4), IN(x+6, y+5), IN(x+6, y+6), IN(x+6, y+7),
//                    IN(x+7, y-7), IN(x+7, y-6), IN(x+7, y-5), IN(x+7, y-4), IN(x+7, y-3), IN(x+7, y-2), IN(x+7, y-1), IN(x+7, y), IN(x+7, y+1), IN(x+7, y+2), IN(x+7, y+3), IN(x+7, y+4), IN(x+7, y+5), IN(x+7, y+6), IN(x+7, y+7)
//        };
//
//        // bubble-sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                { /* swap? */
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
//    }
//
//}


//__global__ void kernel15MS(int nx, int ny, float *d_out, float *d_in)
//{
//    __shared__ float smem[BLOCK_X+14][BLOCK_Y+14];
//
//    int tx = threadIdx.x, ty = threadIdx.y;
//
//    int x = blockIdx.x * blockDim.x + tx;
//    int y = blockIdx.y * blockDim.y + ty;
//
//    bool is_x_top = (tx == 0), is_x_bot = (tx == BLOCK_X-1);
//    bool is_y_top = (ty == 0), is_y_bot = (ty == BLOCK_Y-1);
//
//    if ((x < nx) && (y < ny))
//    {
//        int winSize = 15;
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
////        int roffset = (winSize-1)/2;
////        int toffset = loffset+roffset;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        SMEM(tx , ty) = IN(x, y); // self pixel value
//
//
////        if (is_x_top)
////        {
////            SMEM(tx-1, ty) = IN(x-1, y);
////            SMEM(tx-2, ty) = IN(x-2, y);
////            SMEM(tx-3, ty) = IN(x-3, y);
////            SMEM(tx-4, ty) = IN(x-4, y);
////            SMEM(tx-5, ty) = IN(x-5, y);
////            SMEM(tx-6, ty) = IN(x-6, y);
////            SMEM(tx-7, ty) = IN(x-7, y);
////        }
////        else if (is_x_bot)
////        {
////            SMEM(tx+1, ty) = IN(x+1, y);
////            SMEM(tx+2, ty) = IN(x+2, y);
////            SMEM(tx+3, ty) = IN(x+3, y);
////            SMEM(tx+4, ty) = IN(x+4, y);
////            SMEM(tx+5, ty) = IN(x+5, y);
////            SMEM(tx+6, ty) = IN(x+6, y);
////            SMEM(tx+7, ty) = IN(x+7, y);
////        }
////
////        if (is_y_top)
////        {
////            SMEM(tx, ty-1) = IN(x, y-1);
////            SMEM(tx, ty-2) = IN(x, y-2);
////            SMEM(tx, ty-3) = IN(x, y-3);
////            SMEM(tx, ty-4) = IN(x, y-4);
////            SMEM(tx, ty-5) = IN(x, y-5);
////            SMEM(tx, ty-6) = IN(x, y-6);
////            SMEM(tx, ty-7) = IN(x, y-7);
////            if (is_x_top)
////            {
////                SMEM(tx-1, ty-1) = IN(x-1, y-1);
////                SMEM(tx-2, ty-2) = IN(x-2, y-2);
////                SMEM(tx-3, ty-3) = IN(x-3, y-3);
////                SMEM(tx-4, ty-4) = IN(x-4, y-4);
////                SMEM(tx-5, ty-5) = IN(x-5, y-5);
////                SMEM(tx-6, ty-6) = IN(x-6, y-6);
////                SMEM(tx-7, ty-7) = IN(x-7, y-7);
////
//////                SMEM(tx-1, ty-1) = IN(x-1, y-1);
////
////            }
////            else if (is_x_bot)
////            {
////                SMEM(tx+1, ty-1) = IN(x+1, y-1);
////                SMEM(tx+2, ty-2) = IN(x+2, y-2);
////                SMEM(tx+3, ty-3) = IN(x+3, y-3);
////                SMEM(tx+4, ty-4) = IN(x+4, y-4);
////                SMEM(tx+5, ty-5) = IN(x+5, y-5);
////                SMEM(tx+6, ty-6) = IN(x+6, y-6);
////                SMEM(tx+7, ty-7) = IN(x+7, y-7);
////            }
////        }
////        else if (is_y_bot)
////        {
////            SMEM(tx, ty+1) = IN(x, y+1);
////            SMEM(tx, ty+2) = IN(x, y+2);
////            SMEM(tx, ty+3) = IN(x, y+3);
////            SMEM(tx, ty+4) = IN(x, y+4);
////            SMEM(tx, ty+5) = IN(x, y+5);
////            SMEM(tx, ty+6) = IN(x, y+6);
////            SMEM(tx, ty+7) = IN(x, y+7);
////            if (is_x_top)
////            {
////                SMEM(tx-1, ty+1) = IN(x-1, y+1);
////                SMEM(tx-2, ty+2) = IN(x-2, y+2);
////                SMEM(tx-3, ty+3) = IN(x-3, y+3);
////                SMEM(tx-4, ty+4) = IN(x-4, y+4);
////                SMEM(tx-5, ty+5) = IN(x-5, y+5);
////                SMEM(tx-6, ty+6) = IN(x-6, y+6);
////                SMEM(tx-7, ty+7) = IN(x-7, y+7);
////            }
////            else if (is_x_bot)
////            {
////                SMEM(tx+1, ty+1) = IN(x+1, y+1);
////                SMEM(tx+2, ty+2) = IN(x+2, y+2);
////                SMEM(tx+3, ty+3) = IN(x+3, y+3);
////                SMEM(tx+4, ty+4) = IN(x+4, y+4);
////                SMEM(tx+5, ty+5) = IN(x+5, y+5);
////                SMEM(tx+6, ty+6) = IN(x+6, y+6);
////                SMEM(tx+7, ty+7) = IN(x+7, y+7);
////            }
//
////        }
//    __syncthreads();
////    printf("the x is %d, y is %d, i is , result is %f\n", x, y, SMEM(tx-7, ty-7));
//    // use macro to fetch the value, like loop unrolling
//    float v[225]={SMEM(tx-7, ty-7), SMEM(tx-7, ty-6), SMEM(tx-7, ty-5), SMEM(tx-7, ty-4), SMEM(tx-7, ty-3), SMEM(tx-7, ty-2), SMEM(tx-7, ty-1), SMEM(tx-7, ty), SMEM(tx-7, ty+1), SMEM(tx-7, ty+2), SMEM(tx-7, ty+3), SMEM(tx-7, ty+4), SMEM(tx-7, ty+5), SMEM(tx-7, ty+6), SMEM(tx-7, ty+7),
//                  SMEM(tx-6, ty-7), SMEM(tx-6, ty-6), SMEM(tx-6, ty-5), SMEM(tx-6, ty-4), SMEM(tx-6, ty-3), SMEM(tx-6, ty-2), SMEM(tx-6, ty-1), SMEM(tx-6, ty), SMEM(tx-6, ty+1), SMEM(tx-6, ty+2), SMEM(tx-6, ty+3), SMEM(tx-6, ty+4), SMEM(tx-6, ty+5), SMEM(tx-6, ty+6), SMEM(tx-6, ty+7),
//                  SMEM(tx-5, ty-7), SMEM(tx-5, ty-6), SMEM(tx-5, ty-5), SMEM(tx-5, ty-4), SMEM(tx-5, ty-3), SMEM(tx-5, ty-2), SMEM(tx-5, ty-1), SMEM(tx-5, ty), SMEM(tx-5, ty+1), SMEM(tx-5, ty+2), SMEM(tx-5, ty+3), SMEM(tx-5, ty+4), SMEM(tx-5, ty+5), SMEM(tx-5, ty+6), SMEM(tx-5, ty+7),
//                  SMEM(tx-4, ty-7), SMEM(tx-4, ty-6), SMEM(tx-4, ty-5), SMEM(tx-4, ty-4), SMEM(tx-4, ty-3), SMEM(tx-4, ty-2), SMEM(tx-4, ty-1), SMEM(tx-4, ty), SMEM(tx-4, ty+1), SMEM(tx-4, ty+2), SMEM(tx-4, ty+3), SMEM(tx-4, ty+4), SMEM(tx-4, ty+5), SMEM(tx-4, ty+6), SMEM(tx-4, ty+7),
//                SMEM(tx-3, ty-7), SMEM(tx-3, ty-6), SMEM(tx-3, ty-5), SMEM(tx-3, ty-4), SMEM(tx-3, ty-3), SMEM(tx-3, ty-2), SMEM(tx-3, ty-1), SMEM(tx-3, ty), SMEM(tx-3, ty+1), SMEM(tx-3, ty+2), SMEM(tx-3, ty+3), SMEM(tx-3, ty+4), SMEM(tx-3, ty+5), SMEM(tx-3, ty+6), SMEM(tx-3, ty+7),
//                SMEM(tx-2, ty-7), SMEM(tx-2, ty-6), SMEM(tx-2, ty-5), SMEM(tx-2, ty-4), SMEM(tx-2, ty-3), SMEM(tx-2, ty-2), SMEM(tx-2, ty-1), SMEM(tx-2, ty), SMEM(tx-2, ty+1), SMEM(tx-2, ty+2), SMEM(tx-2, ty+3), SMEM(tx-2, ty+4), SMEM(tx-2, ty+5), SMEM(tx-2, ty+6), SMEM(tx-2, ty+7),
//                SMEM(tx-1, ty-7), SMEM(tx-1, ty-6), SMEM(tx-1, ty-5), SMEM(tx-1, ty-4), SMEM(tx-1, ty-3), SMEM(tx-1, ty-2), SMEM(tx-1, ty-1), SMEM(tx-1, ty), SMEM(tx-1, ty+1), SMEM(tx-1, ty+2), SMEM(tx-1, ty+3), SMEM(tx-1, ty+4), SMEM(tx-1, ty+5), SMEM(tx-1, ty+6), SMEM(tx-1, ty+7),
//                SMEM(tx, ty-7), SMEM(tx, ty-6), SMEM(tx, ty-5), SMEM(tx, ty-4), SMEM(tx, ty-3), SMEM(tx, ty-2), SMEM(tx, ty-1), SMEM(tx, ty), SMEM(tx, ty+1), SMEM(tx, ty+2), SMEM(tx, ty+3), SMEM(tx, ty+4), SMEM(tx, ty+5), SMEM(tx, ty+6), SMEM(tx, ty+7),
//                SMEM(tx+1, ty-7), SMEM(tx+1, ty-6), SMEM(tx+1, ty-5), SMEM(tx+1, ty-4), SMEM(tx+1, ty-3), SMEM(tx+1, ty-2), SMEM(tx+1, ty-1), SMEM(tx+1, ty), SMEM(tx+1, ty+1), SMEM(tx+1, ty+2), SMEM(tx+1, ty+3), SMEM(tx+1, ty+4), SMEM(tx+1, ty+5), SMEM(tx+1, ty+6), SMEM(tx+1, ty+7),
//                SMEM(tx+2, ty-7), SMEM(tx+2, ty-6), SMEM(tx+2, ty-5), SMEM(tx+2, ty-4), SMEM(tx+2, ty-3), SMEM(tx+2, ty-2), SMEM(tx+2, ty-1), SMEM(tx+2, ty), SMEM(tx+2, ty+1), SMEM(tx+2, ty+2), SMEM(tx+2, ty+3), SMEM(tx+2, ty+4), SMEM(tx+2, ty+5), SMEM(tx+2, ty+6), SMEM(tx+2, ty+7),
//                SMEM(tx+3, ty-7), SMEM(tx+3, ty-6), SMEM(tx+3, ty-5), SMEM(tx+3, ty-4), SMEM(tx+3, ty-3), SMEM(tx+3, ty-2), SMEM(tx+3, ty-1), SMEM(tx+3, ty), SMEM(tx+3, ty+1), SMEM(tx+3, ty+2), SMEM(tx+3, ty+3), SMEM(tx+3, ty+4), SMEM(tx+3, ty+5), SMEM(tx+3, ty+6), SMEM(tx+3, ty+7),
//                SMEM(tx+4, ty-7), SMEM(tx+4, ty-6), SMEM(tx+4, ty-5), SMEM(tx+4, ty-4), SMEM(tx+4, ty-3), SMEM(tx+4, ty-2), SMEM(tx+4, ty-1), SMEM(tx+4, ty), SMEM(tx+4, ty+1), SMEM(tx+4, ty+2), SMEM(tx+4, ty+3), SMEM(tx+4, ty+4), SMEM(tx+4, ty+5), SMEM(tx+4, ty+6), SMEM(tx+4, ty+7),
//                SMEM(tx+5, ty-7), SMEM(tx+5, ty-6), SMEM(tx+5, ty-5), SMEM(tx+5, ty-4), SMEM(tx+5, ty-3), SMEM(tx+5, ty-2), SMEM(tx+5, ty-1), SMEM(tx+5, ty), SMEM(tx+5, ty+1), SMEM(tx+5, ty+2), SMEM(tx+5, ty+3), SMEM(tx+5, ty+4), SMEM(tx+5, ty+5), SMEM(tx+5, ty+6), SMEM(tx+5, ty+7),
//                SMEM(tx+6, ty-7), SMEM(tx+6, ty-6), SMEM(tx+6, ty-5), SMEM(tx+6, ty-4), SMEM(tx+6, ty-3), SMEM(tx+6, ty-2), SMEM(tx+6, ty-1), SMEM(tx+6, ty), SMEM(tx+6, ty+1), SMEM(tx+6, ty+2), SMEM(tx+6, ty+3), SMEM(tx+6, ty+4), SMEM(tx+6, ty+5), SMEM(tx+6, ty+6), SMEM(tx+6, ty+7),
//                SMEM(tx+7, ty-7), SMEM(tx+7, ty-6), SMEM(tx+7, ty-5), SMEM(tx+7, ty-4), SMEM(tx+7, ty-3), SMEM(tx+7, ty-2), SMEM(tx+7, ty-1), SMEM(tx+7, ty), SMEM(tx+7, ty+1), SMEM(tx+7, ty+2), SMEM(tx+7, ty+3), SMEM(tx+7, ty+4), SMEM(tx+7, ty+5), SMEM(tx+7, ty+6), SMEM(tx+7, ty+7)
//    };
//
//
//        // bubble-sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                { /* swap? */
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
//    }
//
//}
//
//__global__ void kernel3D2(int nx, int ny, int nz,  float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
//    if ((x < nx) && (y < ny) && (z < nz))
//    {
//        int winSize = 2;
//        float v[4] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//        int newnx=toffset+nx;
//        int newny=toffset+ny;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int yy = y - loffset; yy <= y + roffset; yy++)
//            {
////                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                    v[i++] = d_in[xx+yy*newnx+z*newnx*newny];
//            }
//        }
//
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                { /* swap? */
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];
//
//    }
//
//}

//__global__ void kernel3D15(int nx, int ny, int nz,  float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (y < ny) && (z < nz))
//    {
//        // initial the window size, the local vector size
//        int winSize = 15;
//        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2; // the left and top offset
//        int roffset = (winSize-1)/2; // the right and bottom offset
//        int toffset = loffset+roffset; // the overall offset
//
//// The new x' y' is the plus offset
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//        // Put the neighbour pixel into the local memory for the later bubble sort
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int yy = y - loffset; yy <= y + roffset; yy++)
//            {
//                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//                    // find the read address of the x y z pixel
//                    v[i++] = d_in[xx+yy*(nx+toffset)+z*(nx+toffset)*(ny+toffset)];
//            }
//        }
//
//        // do the bubble sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                {   // bubble sort
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        //    printf("the x is %d, y is %d, z is %d, result is %f \n", x, y, z, v[vecSize/2] );
//        // put the final result value to the output array
//        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];
//
//    }
//
//}
//
//__global__ void kernel3D15XZ(int nx, int ny, int nz,  float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (y < ny) && (z < nz))
//    {
//        // initial the window size, the local vector size
//        int winSize = 15;
//        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2; // the left and top offset
//        int roffset = (winSize-1)/2; // the right and bottom offset
//        int toffset = loffset+roffset; // the overall offset
//
//// The new x' y' is the plus offset
//        x = x + loffset;
////        y = y + loffset;
//        z = z + loffset;
//
//
//        int i = 0;
//        // Put the neighbour pixel into the local memory for the later bubble sort
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int zz = z - loffset; zz <= z + roffset; zz++)
//            {
////                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//                    // find the read address of the x y z pixel
//                    v[i++] = d_in[xx+y*(nx+toffset)+zz*(nx+toffset)*ny];
//            }
//        }
//
//        // do the bubble sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                {   // bubble sort
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
////          printf("the x is %d, y is %d, z is %d, nz is %d  result is %f \n", x, y, z, nz, v[vecSize/2] );
//        // put the final result value to the output array
//        d_out[x-loffset + (z-loffset)*nx + y*nx*nz ] = v[vecSize/2];
//
//    }
//
//}
//
//__global__ void kernel3D15XZME(int nx, int ny, int nz,  float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (y < ny) && (z < nz))
//    {
//        // initial the window size, the local vector size
//        int winSize = 15;
//        float v1[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2; // the left and top offset
//        int roffset = (winSize-1)/2; // the right and bottom offset
//        int toffset = loffset+roffset; // the overall offset
//
//// The new x' y' is the plus offset
//        x = x + loffset;
////        y = y + loffset;
//        z = z + loffset;
//
//// use macro to fetch the value, like loop unrolling
//
//        float v[120]={INZ(x-7,y, z-7), INZ(x-7,y, z-6), INZ(x-7,y, z-5), INZ(x-7,y, z-4), INZ(x-7,y, z-3), INZ(x-7,y, z-2), INZ(x-7,y, z-1), INZ(x-7,y, z), INZ(x-7,y, z+1), INZ(x-7,y, z+2), INZ(x-7,y, z+3), INZ(x-7,y, z+4), INZ(x-7,y, z+5), INZ(x-7,y, z+6), INZ(x-7,y, z+7),
//                      INZ(x-6,y, z-7), INZ(x-6,y, z-6), INZ(x-6,y, z-5), INZ(x-6,y, z-4), INZ(x-6,y, z-3), INZ(x-6,y, z-2), INZ(x-6,y, z-1), INZ(x-6,y, z), INZ(x-6,y, z+1), INZ(x-6,y, z+2), INZ(x-6,y, z+3), INZ(x-6,y, z+4), INZ(x-6,y, z+5), INZ(x-6,y, z+6), INZ(x-6,y, z+7),
//                      INZ(x-5,y, z-7), INZ(x-5,y, z-6), INZ(x-5,y, z-5), INZ(x-5,y, z-4), INZ(x-5,y, z-3), INZ(x-5,y, z-2), INZ(x-5,y, z-1), INZ(x-5,y, z), INZ(x-5,y, z+1), INZ(x-5,y, z+2), INZ(x-5,y, z+3), INZ(x-5,y, z+4), INZ(x-5,y, z+5), INZ(x-5,y, z+6), INZ(x-5,y, z+7),
//                      INZ(x-4,y, z-7), INZ(x-4,y, z-6), INZ(x-4,y, z-5), INZ(x-4,y, z-4), INZ(x-4,y, z-3), INZ(x-4,y, z-2), INZ(x-4,y, z-1), INZ(x-4,y, z), INZ(x-4,y, z+1), INZ(x-4,y, z+2), INZ(x-4,y, z+3), INZ(x-4,y, z+4), INZ(x-4,y, z+5), INZ(x-4,y, z+6), INZ(x-4,y, z+7),
//                    INZ(x-3,y, z-7), INZ(x-3,y, z-6), INZ(x-3,y, z-5), INZ(x-3,y, z-4), INZ(x-3,y, z-3), INZ(x-3,y, z-2), INZ(x-3, y,z-1), INZ(x-3, y,z), INZ(x-3,y, z+1), INZ(x-3,y, z+2), INZ(x-3,y, z+3), INZ(x-3,y, z+4), INZ(x-3,y, z+5), INZ(x-3,y, z+6), INZ(x-3,y, z+7),
//                    INZ(x-2, y,z-7), INZ(x-2,y, z-6), INZ(x-2, y,z-5), INZ(x-2, y,z-4), INZ(x-2,y, z-3), INZ(x-2,y, z-2), INZ(x-2,y, z-1), INZ(x-2,y, z), INZ(x-2,y, z+1), INZ(x-2,y, z+2), INZ(x-2,y, z+3), INZ(x-2,y, z+4), INZ(x-2,y, z+5), INZ(x-2,y, z+6), INZ(x-2,y, z+7),
//                    INZ(x-1,y, z-7), INZ(x-1,y, z-6), INZ(x-1,y, z-5), INZ(x-1,y, z-4), INZ(x-1,y, z-3), INZ(x-1, y,z-2), INZ(x-1,y, z-1), INZ(x-1,y, z), INZ(x-1,y, z+1), INZ(x-1,y, z+2), INZ(x-1,y, z+3), INZ(x-1,y, z+4), INZ(x-1,y, z+5), INZ(x-1,y, z+6), INZ(x-1,y, z+7),
//                    INZ(x, y,z-7), INZ(x, y,z-6), INZ(x,y, z-5), INZ(x,y, z-4), INZ(x, y,z-3), INZ(x,y, z-2), INZ(x,y, z-1), INZ(x,y, z), INZ(x, y,z+1), INZ(x,y, z+2), INZ(x,y, z+3), INZ(x,y, z+4), INZ(x,y, z+5), INZ(x,y, z+6), INZ(x,y, z+7)
//                    };
//
////        int i = 0;
////        // Put the neighbour pixel into the local memory for the later bubble sort
////        for (int xx = x - loffset; xx <= x + roffset; xx++)
////        {
////            for (int zz = z - loffset; zz <= z + roffset; zz++)
////            {
////                    v1[i++] = d_in[xx+y*(nx+toffset)+zz*(nx+toffset)*ny];
////            }
////        }
//
//
////        for (int i = 0; i< 120; i++)
////        {
////
////                printf("the x is %d, y is %d, z is %d, v is %f, v[1] is %f \n", x, y, z, v[i], v1[i] );
////                if(v[i] != v1[i])
////                    printf("the false \n");
////
////        }
//        // do the bubble sort
////        for (int i = 0; i < vecSize; i++)
////        {
////            for (int j = i + 1; j < vecSize; j++)
////            {
////                if (v[i] > v[j])
////                {   // bubble sort
////                    float tmp = v[i];
////                    v[i] = v[j];
////                    v[j] = tmp;
////                }
////            }
////        }
//
//
//        const int ARR_SIZE = 120;
//
//#pragma unroll
//        for(int i = 0; i < ARR_SIZE/2; i++) {
//            swapd(v[i], v[ARR_SIZE-1-i]);
//        }
//
//#pragma unroll
//        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
//            swapd(v[0], v[i]);
//        }
//
//#pragma unroll
//        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
//            swapd(v[i], v[ARR_SIZE-1]);
//        }
//
//        int last = ARR_SIZE-1;
//
//        for(int k = 1; k <= 7; k++) {
//
//            for(int j = -7; j <= 7; j++) {
//
//                // add new contestant to first position in array
//                v[0] = INZ(x+k, y, z+j);
//
////                IN(x+k, y+j);
//
//                last--;
//
//                // place max in last half, min in first half
//                for(int i = 0; i < (last+1)/2; i++) {
//                    swapd(v[i], v[last-i]);
//                }
//                // now perform swaps on each half such that
//                // max is in last pos, min is in first pos
//                for(int i = 1; i <= last/2; i++) {
//                    swapd(v[0], v[i]);
//                }
//                for(int i = last-1; i >= (last+1)/2; i--) {
//                    swapd(v[i], v[last]);
//                }
//            }
//        }
//
//        for(int k = 1; k < 7; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < 7; i++) {
//                swapd(v[i], v[15-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= 7; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = 15-k-2; i >= 7; i--) {
//                swapd(v[i], v[15-1-k]);
//            }
//        }
//
//        for(int k = 1; k < 7; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < 7; i++) {
//                swapd(v[i], v[15-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= 7; i++) {
//                swapd(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = 15-k-2; i >= 7; i--) {
//                swapd(v[i], v[15-1-k]);
//            }
//        }
//
//
////          printf("the x is %d, y is %d, z is %d, nz is %d  result is %f \n", x, y, z, nz, v[vecSize/2] );
//        // put the final result value to the output array
//        d_out[x-loffset + (z-loffset)*nx + y*nx*nz ] = v[7];
//
//    }
//
//}
//
//__global__ void kernelLool3D15(int nx, int ny, int nz,  float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
////    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (y < ny))
//    {
//        int winSize = 15;
//        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//
//
//
//        for(int z = 0; z < nz; z++)
//        {
//            i = 0;
//
//            for (int xx = x - loffset; xx <= x + roffset; xx++)
//            {
//                for (int yy = y - loffset; yy <= y + roffset; yy++)
//                {
//
//                    v[i++] = d_in[xx+yy*(nx+toffset)+z*(nx+toffset)*(ny+toffset)];
//                }
//            }
//
//            for (int i = 0; i < vecSize; i++)
//            {
//                for (int j = i + 1; j < vecSize; j++)
//                {
//                    if (v[i] > v[j])
//                    { /* swap? */
//                        float tmp = v[i];
//                        v[i] = v[j];
//                        v[j] = tmp;
//                    }
//                }
//            }
//
////            printf("the x is %d, y is %d, z is %d, result is %f \n", x, y, z, v[vecSize/2] );
//
//            d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];
//
//        }
//
//    }
//
//}
//
//__global__ void kernelLool3D15XZY(int nx, int ny, int nz,  float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned int z = blockIdx.y*blockDim.y + threadIdx.y;
////    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (z < nz))
//    {
//        int winSize = 15;
//        float v[225]={0};
//
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//
//        int newnx = nx+toffset;
//        int newny = ny+toffset;
//        int zoffset = z*newnx*newny;
//
//        x = x + loffset;
////        y = y + loffset;
////        z = z + loffset;
//
//        int i = 0;
//
//        for(int y = loffset; y < ny+loffset; y++)
//        {
//            i = 0;
//
////            for (int xx = x - loffset; xx <= x + roffset; xx++)
////            {
////                for (int yy = y - loffset; yy <= y + roffset; yy++)
////                {
////
////                    v[i++] = d_in[xx+yy*newnx+zoffset];
////                }
////            }
//
//
//
//            for (int i = 0; i < vecSize; i++)
//            {
//                for (int j = i + 1; j < vecSize; j++)
//                {
//                    if (v[i] > v[j])
//                    {
//                        float tmp = v[i];
//                        v[i] = v[j];
//                        v[j] = tmp;
//                    }
//                }
//            }
//
////            printf("the x is %d, y is %d, z is %d, result is %f \n", x, y, z, v[vecSize/2] );
//
//            d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2];
//
//        }
//
//    }
//
//}
//
//__global__ void reomveOutliner3D2(int nx, int ny, int nz, int diff, float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (y < ny) && (z < nz))
//    {
//        int winSize = 2;
//        float v[4] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//        int newnx=toffset+nx;
//        int newny=toffset+ny;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int yy = y - loffset; yy <= y + roffset; yy++)
//            {
////                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                    v[i++] = d_in[xx+yy*newnx+z*newnx*newny];
//            }
//        }
//
//        // get the current pixel value
//        // TODO get from local buffer instead of global memory
//
//        float currentPixel = d_in[x+y*newnx+z*newnx*newny];
//
//
//
//        // More optimize for the bubble sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                { /* swap? */
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        // TODO more optimize for this part
//        int mask = 0;
//        float realdiff = currentPixel-v[vecSize/2];
//        printf("the x is %d, y is %d, z is %d, current is %f, result is %f \n", x, y, z, currentPixel, v[vecSize/2] );
//
//        if( realdiff >= diff)
//            mask = 1;
//        else
//            mask = 0;
//
//
//
//        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2]*mask+currentPixel*(1-mask);
//
//    }
//
//}

//__global__ void reomveOutliner2D15(int nx, int ny, int diff, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if ((x < nx) && (y < ny))
//    {
//        int winSize = 15;
//        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int yy = y - loffset; yy <= y + roffset; yy++)
//            {
//                v[i++] = d_in[yy*(nx+toffset) + xx];
//            }
//        }
//
//        float currentPixel = IN(x, y);
//
//        // bubble-sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                {
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        int mask = 0;
//        if((currentPixel-v[vecSize/2]) >= diff)
//            mask = 1;
//        else
//            mask = 0;
//
//        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2]*mask+currentPixel*(1-mask);
//    }
//
//}



//__global__ void reomveOutliner2D15M(int nx, int ny, int diff, float *d_out, float *d_in)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if ((x < nx) && (y < ny))
//    {
//        int winSize = 15;
////        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
////        int toffset = loffset+roffset;
//
//        x = x + loffset;
//        y = y + loffset;
//
////        int i = 0;
//
//        float v[225]={IN(x-7, y-7), IN(x-7, y-6), IN(x-7, y-5), IN(x-7, y-4), IN(x-7, y-3), IN(x-7, y-2), IN(x-7, y-1), IN(x-7, y), IN(x-7, y+1), IN(x-7, y+2), IN(x-7, y+3), IN(x-7, y+4), IN(x-7, y+5), IN(x-7, y+6), IN(x-7, y+7),
//                      IN(x-6, y-7), IN(x-6, y-6), IN(x-6, y-5), IN(x-6, y-4), IN(x-6, y-3), IN(x-6, y-2), IN(x-6, y-1), IN(x-6, y), IN(x-6, y+1), IN(x-6, y+2), IN(x-6, y+3), IN(x-6, y+4), IN(x-6, y+5), IN(x-6, y+6), IN(x-6, y+7),
//                      IN(x-5, y-7), IN(x-5, y-6), IN(x-5, y-5), IN(x-5, y-4), IN(x-5, y-3), IN(x-5, y-2), IN(x-5, y-1), IN(x-5, y), IN(x-5, y+1), IN(x-5, y+2), IN(x-5, y+3), IN(x-5, y+4), IN(x-5, y+5), IN(x-5, y+6), IN(x-5, y+7),
//                      IN(x-4, y-7), IN(x-4, y-6), IN(x-4, y-5), IN(x-4, y-4), IN(x-4, y-3), IN(x-4, y-2), IN(x-4, y-1), IN(x-4, y), IN(x-4, y+1), IN(x-4, y+2), IN(x-4, y+3), IN(x-4, y+4), IN(x-4, y+5), IN(x-4, y+6), IN(x-4, y+7),
//                    IN(x-3, y-7), IN(x-3, y-6), IN(x-3, y-5), IN(x-3, y-4), IN(x-3, y-3), IN(x-3, y-2), IN(x-3, y-1), IN(x-3, y), IN(x-3, y+1), IN(x-3, y+2), IN(x-3, y+3), IN(x-3, y+4), IN(x-3, y+5), IN(x-3, y+6), IN(x-3, y+7),
//                    IN(x-2, y-7), IN(x-2, y-6), IN(x-2, y-5), IN(x-2, y-4), IN(x-2, y-3), IN(x-2, y-2), IN(x-2, y-1), IN(x-2, y), IN(x-2, y+1), IN(x-2, y+2), IN(x-2, y+3), IN(x-2, y+4), IN(x-2, y+5), IN(x-2, y+6), IN(x-2, y+7),
//                    IN(x-1, y-7), IN(x-1, y-6), IN(x-1, y-5), IN(x-1, y-4), IN(x-1, y-3), IN(x-1, y-2), IN(x-1, y-1), IN(x-1, y), IN(x-1, y+1), IN(x-1, y+2), IN(x-1, y+3), IN(x-1, y+4), IN(x-1, y+5), IN(x-1, y+6), IN(x-1, y+7),
//                    IN(x, y-7), IN(x, y-6), IN(x, y-5), IN(x, y-4), IN(x, y-3), IN(x, y-2), IN(x, y-1), IN(x, y), IN(x, y+1), IN(x, y+2), IN(x, y+3), IN(x, y+4), IN(x, y+5), IN(x, y+6), IN(x, y+7),
//                    IN(x+1, y-7), IN(x+1, y-6), IN(x+1, y-5), IN(x+1, y-4), IN(x+1, y-3), IN(x+1, y-2), IN(x+1, y-1), IN(x+1, y), IN(x+1, y+1), IN(x+1, y+2), IN(x+1, y+3), IN(x+1, y+4), IN(x+1, y+5), IN(x+1, y+6), IN(x+1, y+7),
//                    IN(x+2, y-7), IN(x+2, y-6), IN(x+2, y-5), IN(x+2, y-4), IN(x+2, y-3), IN(x+2, y-2), IN(x+2, y-1), IN(x+2, y), IN(x+2, y+1), IN(x+2, y+2), IN(x+2, y+3), IN(x+2, y+4), IN(x+2, y+5), IN(x+2, y+6), IN(x+2, y+7),
//                    IN(x+3, y-7), IN(x+3, y-6), IN(x+3, y-5), IN(x+3, y-4), IN(x+3, y-3), IN(x+3, y-2), IN(x+3, y-1), IN(x+3, y), IN(x+3, y+1), IN(x+3, y+2), IN(x+3, y+3), IN(x+3, y+4), IN(x+3, y+5), IN(x+3, y+6), IN(x+3, y+7),
//                    IN(x+4, y-7), IN(x+4, y-6), IN(x+4, y-5), IN(x+4, y-4), IN(x+4, y-3), IN(x+4, y-2), IN(x+4, y-1), IN(x+4, y), IN(x+4, y+1), IN(x+4, y+2), IN(x+4, y+3), IN(x+4, y+4), IN(x+4, y+5), IN(x+4, y+6), IN(x+4, y+7),
//                    IN(x+5, y-7), IN(x+5, y-6), IN(x+5, y-5), IN(x+5, y-4), IN(x+5, y-3), IN(x+5, y-2), IN(x+5, y-1), IN(x+5, y), IN(x+5, y+1), IN(x+5, y+2), IN(x+5, y+3), IN(x+5, y+4), IN(x+5, y+5), IN(x+5, y+6), IN(x+5, y+7),
//                    IN(x+6, y-7), IN(x+6, y-6), IN(x+6, y-5), IN(x+6, y-4), IN(x+6, y-3), IN(x+6, y-2), IN(x+6, y-1), IN(x+6, y), IN(x+6, y+1), IN(x+6, y+2), IN(x+6, y+3), IN(x+6, y+4), IN(x+6, y+5), IN(x+6, y+6), IN(x+6, y+7),
//                    IN(x+7, y-7), IN(x+7, y-6), IN(x+7, y-5), IN(x+7, y-4), IN(x+7, y-3), IN(x+7, y-2), IN(x+7, y-1), IN(x+7, y), IN(x+7, y+1), IN(x+7, y+2), IN(x+7, y+3), IN(x+7, y+4), IN(x+7, y+5), IN(x+7, y+6), IN(x+7, y+7)
//        };
//
//        float currentPixel = IN(x, y);
//
//        // bubble-sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                {
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        int mask = 0;
//        if((currentPixel-v[vecSize/2]) >= diff)
//            mask = 1;
//        else
//            mask = 0;
//
//        // pick the middle one
//        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2]*mask+currentPixel*(1-mask);
//    }
//
//}

//
//
//
//__global__ void reomveOutliner3D15(int nx, int ny, int nz, int diff, float *d_out, float *d_in)
//{
//   // nx ny nz map to offset in the 1d array
//    unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
//    unsigned z = blockIdx.z*blockDim.z + threadIdx.z;
//
////    int offset = x+y* nx + ny * nx * z;
//    if ((x < nx) && (y < ny) && (z < nz))
//    {
//        int winSize = 15;
//        float v[225] = {0};
//
//        int vecSize = winSize*winSize;
//        int loffset = winSize/2;
//        int roffset = (winSize-1)/2;
//        int toffset = loffset+roffset;
//        int newnx=toffset+nx;
//        int newny=toffset+ny;
//
//        x = x + loffset;
//        y = y + loffset;
//
//        int i = 0;
//
//        for (int xx = x - loffset; xx <= x + roffset; xx++)
//        {
//            for (int yy = y - loffset; yy <= y + roffset; yy++)
//            {
////                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries
//
//                    v[i++] = d_in[xx+yy*newnx+z*newnx*newny];
//            }
//        }
//
//        // get the current pixel value
//        // TODO get from local buffer instead of global memory
//
//        float currentPixel = d_in[x+y*newnx+z*newnx*newny];
//
////        printf("the x is %d, y is %d, z is %d, current is %f, result is %f \n", x, y, z, currentPixel, v[vecSize/2] );
//
//        // More optimize for the bubble sort
//        for (int i = 0; i < vecSize; i++)
//        {
//            for (int j = i + 1; j < vecSize; j++)
//            {
//                if (v[i] > v[j])
//                { /* swap? */
//                    float tmp = v[i];
//                    v[i] = v[j];
//                    v[j] = tmp;
//                }
//            }
//        }
//
//        // TODO more optimize for this part
//        int mask = 0;
//        if((currentPixel-v[vecSize/2]) >= diff)
//            mask = 1;
//        else
//            mask = 0;
//
//
//
//        d_out[x-loffset + (y-loffset)*nx + z*nx*ny ] = v[vecSize/2]*mask+currentPixel*(1-mask);
//
//    }
//
//}



