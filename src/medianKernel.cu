#include "hip/hip_runtime.h"
// Only support reflect mode right now

#include <stdio.h>
#include <medianFilter.hh>


// various windows size
//__global__ void kernel(int nx, int ny, float *d_out, float *d_in, int size)
//{
//
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    int i = 0;
////    float v[9] = {0,0,0,0,0,0,0,0,0}; // zero padding
//
//    int offset = (size-1)/2;
//    int winSize = size*size;
//    int winOffset = (winSize-1)/2;
//
//    float v[winSize] = {0};
//
//
//    for (int xx = x - offset; xx <= x + offset; xx++) {
//        for (int yy = y - offset; yy <= y + offset; yy++) {
//            if (0 <= xx && xx < nx && 0 <= yy && yy < ny) // boundaries
//
//                v[i++] = d_in[yy*nx + xx];
//        }
//    }
//
//    // bubble-sort
//    for (int i = 0; i < winSize; i++) {
//        for (int j = i + 1; j < winSize; j++) {
//            if (v[i] > v[j]) { /* swap? */
//                float tmp = v[i];
//                v[i] = v[j];
//                v[j] = tmp;
//            }
//        }
//    }
//
////     printf("the x not is %d, y is %d, result is %f \n", x, y, v[4] );
//    // pick the middle one
//    d_out[y*nx + x] = v[winOffset];
//}


// window 2 by 2
__global__ void kernel2(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;



        int winSize = 2;
        float v[4] = {0};

        int vecSize = winSize*winSize;
        int loffset = winSize/2;
        int roffset = winSize/2 - 1;
        int toffset = loffset+roffset;

        x = x + loffset;
        y = y + loffset;



        int i = 0;


        for (int xx = x - loffset; xx <= x + roffset; xx++)
        {
            for (int yy = y - loffset; yy <= y + roffset; yy++)
            {
                if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset)
                 {// boundaries
                    v[i++] = d_in[yy*(nx+toffset) + xx];
                 }
            }
        }

        // bubble-sort
        for (int i = 0; i < vecSize; i++)
        {
            for (int j = i + 1; j < vecSize; j++)
            {
                if (v[i] > v[j])
                { /* swap? */
                    float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }

        // pick the middle one
        d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];



}


// window 3 by 3
__global__ void kernel3(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 3;
    float v[9] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = (winSize-1)/2;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
}

// windows size 4 byb 4
__global__ void kernel4(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 4;
    int loffset = winSize/2;
    int roffset = winSize/2 - 1;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;
    float v[16] = {0};

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < 16; i++)
    {
        for (int j = i + 1; j < 16; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[8];
}

// Windows size 5 by b5
__global__ void kernel5(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 5;
    float v[25] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = (winSize-1)/2;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];

}

// windows size 6 byb 6
__global__ void kernel6(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 6;
    float v[36] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = winSize/2 - 1;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;


    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];
}


// window size 15 by b15
__global__ void kernel15(int nx, int ny, float *d_out, float *d_in)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winSize = 15;
    float v[225] = {0};

    int vecSize = winSize*winSize;
    int loffset = winSize/2;
    int roffset = (winSize-1)/2;
    int toffset = loffset+roffset;

    x = x + loffset;
    y = y + loffset;

    int i = 0;

    for (int xx = x - loffset; xx <= x + roffset; xx++)
    {
        for (int yy = y - loffset; yy <= y + roffset; yy++)
        {
            if (0 <= xx && xx < nx+toffset && 0 <= yy && yy < ny+toffset) // boundaries

                v[i++] = d_in[yy*(nx+toffset) + xx];
        }
    }

    // bubble-sort
    for (int i = 0; i < vecSize; i++)
    {
        for (int j = i + 1; j < vecSize; j++)
        {
            if (v[i] > v[j])
            { /* swap? */
                float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }

    // pick the middle one
    d_out[(y-loffset)*nx + x-loffset] = v[vecSize/2];

}

//#define s2(a,b)            { float tmp = a; a = min(a,b); b = max(tmp,b); }
//#define mn3(a,b,c)         s2(a,b); s2(a,c);
//#define mx3(a,b,c)         s2(b,c); s2(a,c);
//
//#define mnmx3(a,b,c)       mx3(a,b,c); s2(a,b);                               // 3 exchanges
//#define mnmx4(a,b,c,d)     s2(a,b); s2(c,d); s2(a,c); s2(b,d);                // 4 exchanges
//#define mnmx5(a,b,c,d,e)   s2(a,b); s2(c,d); mn3(a,c,e); mx3(b,d,e);          // 6 exchanges
//#define mnmx6(a,b,c,d,e,f) s2(a,d); s2(b,e); s2(c,f); mn3(a,b,c); mx3(d,e,f); // 7 exchanges
//
//#define SMEM(x,y)  smem[(x)+1][(y)+1]
//#define IN(x,y)    d_in[(y)*nx + (x)]
//
// __global__ void kernel(int nx, int ny, float *d_out, float *d_in, int size)
//{
//
//    int tx = threadIdx.x, ty = threadIdx.y;
//
//    // guards: is at boundary?
//    bool is_x_top = (tx == 0), is_x_bot = (tx == BLOCK_X-1);
//    bool is_y_top = (ty == 0), is_y_bot = (ty == BLOCK_Y-1);
//
//    __shared__ float smem[BLOCK_X+2][BLOCK_Y+2];
//    // clear out shared memory (zero padding)
//    if (is_x_top)           SMEM(tx-1, ty  ) = 0;
//    else if (is_x_bot)      SMEM(tx+1, ty  ) = 0;
//    if (is_y_top) {         SMEM(tx  , ty-1) = 0;
//        if (is_x_top)       SMEM(tx-1, ty-1) = 0;
//        else if (is_x_bot)  SMEM(tx+1, ty-1) = 0;
//    } else if (is_y_bot) {  SMEM(tx  , ty+1) = 0;
//        if (is_x_top)       SMEM(tx-1, ty+1) = 0;
//        else if (is_x_bot)  SMEM(tx+1, ty+1) = 0;
//    }
//
//    // guards: is at boundary and still more image?
//    int x = blockIdx.x * blockDim.x + tx;
//    int y = blockIdx.y * blockDim.y + ty;
//    is_x_top &= (x > 0); is_x_bot &= (x < nx - 1);
//    is_y_top &= (y > 0); is_y_bot &= (y < ny - 1);
//
//    // each thread pulls from image
//                            SMEM(tx  , ty  ) = IN(x  , y  ); // self
//    if (is_x_top)           SMEM(tx-1, ty  ) = IN(x-1, y  );
//    else if (is_x_bot)      SMEM(tx+1, ty  ) = IN(x+1, y  );
//    if (is_y_top) {         SMEM(tx  , ty-1) = IN(x  , y-1);
//        if (is_x_top)       SMEM(tx-1, ty-1) = IN(x-1, y-1);
//        else if (is_x_bot)  SMEM(tx+1, ty-1) = IN(x+1, y-1);
//    } else if (is_y_bot) {  SMEM(tx  , ty+1) = IN(x  , y+1);
//        if (is_x_top)       SMEM(tx-1, ty+1) = IN(x-1, y+1);
//        else if (is_x_bot)  SMEM(tx+1, ty+1) = IN(x+1, y+1);
//    }
//    __syncthreads();
//
//    // pull top six from shared memory
//    float v[6] = { SMEM(tx-1, ty-1), SMEM(tx  , ty-1), SMEM(tx+1, ty-1),
//                   SMEM(tx-1, ty  ), SMEM(tx  , ty  ), SMEM(tx+1, ty  ) };
//
//    // with each pass, remove min and max values and add new value
//    mnmx6(v[0], v[1], v[2], v[3], v[4], v[5]);
//    v[5] = SMEM(tx-1, ty+1); // add new contestant
//    mnmx5(v[1], v[2], v[3], v[4], v[5]);
//    v[5] = SMEM(tx  , ty+1);
//    mnmx4(v[2], v[3], v[4], v[5]);
//    v[5] = SMEM(tx+1, ty+1);
//    mnmx3(v[3], v[4], v[5]);
//
////    printf("the x is %d, y is %d, result is %f \n", x, y, v[4] );
//
//    // pick the middle one
//    d_out[y*nx + x] = v[4];
//}
