
#include <hip/hip_runtime.h>
//
//static const int THREADS_X = 16;
//static const int THREADS_Y = 16;
//
//#define __DH__ __device__ __host__
//
//typedef long long   dim_t;
//
//// Exchange trick: Morgan McGuire, ShaderX 2008
//#define swap(a,b)    { T tmp = a; a = min(a,b); b = max(tmp,b); }
//
//typedef enum {
//    ///
//    /// Out of bound values are 0
//    ///
//    AF_PAD_ZERO = 0,
//
//    ///
//    /// Out of bound values are symmetric over the edge
//    ///
//    AF_PAD_SYM
//} af_border_type;
//
//
//template<typename T>
//struct Param
//{
//    T *ptr;
//    dim_t dims[4];
//    dim_t strides[4];
//};
//
//template<typename T>
//class CParam
//{
//public:
//    const T *ptr;
//    dim_t dims[4];
//    dim_t strides[4];
//
//    __DH__ CParam(const T *iptr, const dim_t *idims, const dim_t *istrides) :
//        ptr(iptr)
//    {
//        for (int i = 0; i < 4; i++) {
//            dims[i] = idims[i];
//            strides[i] = istrides[i];
//        }
//    }
//
//    __DH__ CParam(Param<T> &in) : ptr(in.ptr)
//    {
//        for (int i = 0; i < 4; i++) {
//            dims[i] = in.dims[i];
//            strides[i] = in.strides[i];
//        }
//    }
//
//    __DH__ ~CParam() {}
//};
//
//__forceinline__ __device__
//int lIdx(int x, int y, int stride1, int stride0)
//{
//    return (y*stride1 + x*stride0);
//}
//
//template<typename T, af_border_type pad>
//__device__
//void load2ShrdMem(T * shrd, const T * in,
//                  int lx, int ly, int shrdStride,
//                  int dim0, int dim1,
//                  int gx, int gy,
//                  int inStride1, int inStride0)
//{
//    switch(pad) {
//        case AF_PAD_ZERO:
//            {
//                if (gx<0 || gx>=dim0 || gy<0 || gy>=dim1)
//                    shrd[lIdx(lx, ly, shrdStride, 1)] = T(0);
//                else
//                    shrd[lIdx(lx, ly, shrdStride, 1)] = in[lIdx(gx, gy, inStride1, inStride0)];
//            }
//            break;
//        case AF_PAD_SYM:
//            {
//                if (gx<0) gx *= -1;
//                if (gy<0) gy *= -1;
//                if (gx>=dim0) gx = 2*(dim0-1) - gx;
//                if (gy>=dim1) gy = 2*(dim1-1) - gy;
//
//                shrd[lIdx(lx, ly, shrdStride, 1)] = in[lIdx(gx, gy, inStride1, inStride0)];
//            }
//            break;
//    }
//}
//
//template<typename T, af_border_type pad, unsigned w_len, unsigned w_wid>
//__global__
//void medfilt(Param<T> out, CParam<T> in, int nBBS0, int nBBS1)
//{
//    __shared__ T shrdMem[(THREADS_X+w_len-1)*(THREADS_Y+w_wid-1)];
//
//    // calculate necessary offset and window parameters
//    const int padding = w_len-1;
//    const int halo    = padding/2;
//    const int shrdLen = blockDim.x + padding;
//
//    // batch offsets
//    unsigned b2 = blockIdx.x / nBBS0;
//    unsigned b3 = blockIdx.y / nBBS1;
//    const T* iptr    = (const T *) in.ptr + (b2 *  in.strides[2] + b3 *  in.strides[3]);
//    T*       optr    = (T *      )out.ptr + (b2 * out.strides[2] + b3 * out.strides[3]);
//
//    // local neighborhood indices
//    int lx = threadIdx.x;
//    int ly = threadIdx.y;
//
//    // global indices
//    int gx = blockDim.x * (blockIdx.x-b2*nBBS0) + lx;
//    int gy = blockDim.y * (blockIdx.y-b3*nBBS1) + ly;
//
//    // pull image to local memory
//    for (int b=ly, gy2=gy; b<shrdLen; b+=blockDim.y, gy2+=blockDim.y) {
//        // move row_set get_local_size(1) along coloumns
//        for (int a=lx, gx2=gx; a<shrdLen; a+=blockDim.x, gx2+=blockDim.x) {
//            load2ShrdMem<T, pad>(shrdMem, iptr, a, b, shrdLen, in.dims[0], in.dims[1],
//                    gx2-halo, gy2-halo, in.strides[1], in.strides[0]);
//        }
//    }
//
//    __syncthreads();
//
//    // Only continue if we're at a valid location
//    if (gx < in.dims[0] && gy < in.dims[1]) {
//
//        const int ARR_SIZE = w_len * (w_wid-w_wid/2);
//        // pull top half from shared memory into local memory
//        T v[ARR_SIZE];
//#pragma unroll
//        for(int k = 0; k <= w_wid/2; k++) {
//#pragma unroll
//            for(int i = 0; i < w_len; i++) {
//                v[w_len*k + i] = shrdMem[lIdx(lx+i,ly+k,shrdLen,1)];
//            }
//        }
//
//        // with each pass, remove min and max values and add new value
//        // initial sort
//        // ensure min in first half, max in second half
//#pragma unroll
//        for(int i = 0; i < ARR_SIZE/2; i++) {
//            swap(v[i], v[ARR_SIZE-1-i]);
//        }
//        // move min in first half to first pos
//#pragma unroll
//        for(int i = 1; i < (ARR_SIZE+1)/2; i++) {
//            swap(v[0], v[i]);
//        }
//        // move max in second half to last pos
//#pragma unroll
//        for(int i = ARR_SIZE-2; i >= ARR_SIZE/2; i--) {
//            swap(v[i], v[ARR_SIZE-1]);
//        }
//
//        int last = ARR_SIZE-1;
//
//        for(int k = 1+w_wid/2; k < w_wid; k++) {
//
//            for(int j = 0; j < w_len; j++) {
//
//                // add new contestant to first position in array
//                v[0] = shrdMem[lIdx(lx+j, ly+k, shrdLen, 1)];
//
//                last--;
//
//                // place max in last half, min in first half
//                for(int i = 0; i < (last+1)/2; i++) {
//                    swap(v[i], v[last-i]);
//                }
//                // now perform swaps on each half such that
//                // max is in last pos, min is in first pos
//                for(int i = 1; i <= last/2; i++) {
//                    swap(v[0], v[i]);
//                }
//                for(int i = last-1; i >= (last+1)/2; i--) {
//                    swap(v[i], v[last]);
//                }
//            }
//        }
//
//        // no more new contestants
//        // may still have to sort the last row
//        // each outer loop drops the min and max
//        for(int k = 1; k < w_len/2; k++) {
//            // move max/min into respective halves
//            for(int i = k; i < w_len/2; i++) {
//                swap(v[i], v[w_len-1-i]);
//            }
//            // move min into first pos
//            for(int i = k+1; i <= w_len/2; i++) {
//                swap(v[k], v[i]);
//            }
//            // move max into last pos
//            for(int i = w_len-k-2; i >= w_len/2; i--) {
//                swap(v[i], v[w_len-1-k]);
//            }
//        }
//
//        // pick the middle element of the first row
//        optr[gy*out.strides[1]+gx*out.strides[0]] = v[w_len/2];
//    }
//}