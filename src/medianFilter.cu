#include <medianKernel.cu>
#include <medianFilter.hh>
#include <assert.h>
//#include <time.h>
#include <sys/time.h>
#include <iostream>


using namespace std;


medianFilter::medianFilter (/*float* array_host_,*/ int nx_, int ny_, int nz_, int filterSize_)
{

// the matrix size
  nx = nx_;
  ny = ny_;
  nz = nz_;

  filterSize = filterSize_;

  // inlength is the 2d image size of each image
  // outlength is the 2d image size of each image
//  inlength=(nx+filterSize-1)*(ny+filterSize-1)*nz;
//  outlength = nx*ny*nz;

  // inlength is the 2d image size of each image
  inlength=(nx+filterSize-1)*(ny+filterSize-1);
  outlength = nx*ny;


//  for (int i =0; i< inlength; i++)
//    printf("the element is %f \n", array_host[i]);

  insize = inlength * sizeof(float);
  outsize = outlength * sizeof(float);

//  if ((array_host_out = (float*)malloc(sizeof(float)*length))) == 0)
//  {
//    fprintf(stderr,"malloc1 Fail \n");
//    return;
//  }

  hipError_t err = hipMalloc((void**) &array_device_in, insize);
  assert(err == 0);

  err = hipMalloc((void**) &array_device_out, outsize);
  assert(err == 0);

//  err = hipMemcpy(array_device_in, array_host, insize, hipMemcpyHostToDevice);
//  assert(err == 0);



}

medianFilter::~medianFilter()
{

//  hipFree(v_device);

  hipFree(array_device_in);
  hipFree(array_device_out);

}

void medianFilter::run2DFilter(int size)
{

//  hipError_t err0 = hipMalloc((void**) &v_device, size*size*sizeof(float));
//  hipMemset(v_device, 0, size*size*sizeof(float));
//  double iStart = cpuSecond();

  int block_size_x = BLOCK_X;
  int block_size_y = BLOCK_Y;

  dim3 blocks((nx+block_size_x-1)/block_size_x, (ny+block_size_y-1)/block_size_y);
  dim3 threads(block_size_x,block_size_y);

//  af_border_type pad = AF_PAD_SYM;
//  Param<float> out;
//  out.ptr=array_device_out;
//  out.dims[0]=


  switch(filterSize)
  {
    case 2:
      kernel2ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 3:
      kernel3ME <<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 4:
      kernel4ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 5:
      kernel5ME <<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 6:
      kernel6ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 7:
      kernel7ME <<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 8:
      kernel8ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 9:
      kernel9ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 10:
      kernel10ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 11:
      kernel11ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 12:
      kernel12ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 13:
      kernel13ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 14:
      kernel14ME<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 15:
      kernel15ME <<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    default:
      break;

  }

  // add these to synchronzie the thread
  hipDeviceSynchronize();

//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));

//  hipError_t err = hipGetLastError();
//  assert(err == 0);
//  gpuErrchk( hipPeekAtLastError());
//  gpuErrchk( hipDeviceSynchronize() );

}

void medianFilter::run2DRemoveOutliner(int size, int diff)
{



//  double iStart = cpuSecond();

  int block_size_x = BLOCK_X;
  int block_size_y = BLOCK_Y;

  dim3 blocks((nx+block_size_x-1)/block_size_x, (ny+block_size_y-1)/block_size_y);
  dim3 threads(block_size_x,block_size_y);


  switch(filterSize)
  {
    case 2:
      reomveOutliner2D2ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 3:
      reomveOutliner2D3ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 4:
      reomveOutliner2D4ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 5:
      reomveOutliner2D5ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 6:
      reomveOutliner2D6ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 7:
      reomveOutliner2D7ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 8:
      reomveOutliner2D8ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 9:
      reomveOutliner2D9ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 10:
      reomveOutliner2D10ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 11:
      reomveOutliner2D11ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 12:
      reomveOutliner2D12ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 13:
      reomveOutliner2D13ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 14:
      reomveOutliner2D14ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    case 15:
//      reomveOutliner2D15<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
//      reomveOutliner2D15M<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      reomveOutliner2D15ME<<<blocks,threads>>>(nx, ny, diff, array_device_out, array_device_in);
      break;
    default:
      break;

  }


  // add these to synchronzie the thread
  hipDeviceSynchronize();

//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));

  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void medianFilter::retreive()
{

//  int outsize = outlength * sizeof(float);
  hipMemcpy(array_host, array_device_out, outsize, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0)
  {
    cout << err << endl; assert(0);
  }

}

void medianFilter::retreive_to (float* array_host_)
{
//  assert(length == length_);
//  time_t start = time(NULL);

//  double iStart = cpuSecond();

  hipMemcpy(array_host_, array_device_out, outsize, hipMemcpyDeviceToHost);

//  printf("total copy back time for this process took %f sec \n",(cpuSecond() - iStart));

  hipError_t err = hipGetLastError();
  assert(err == 0);
}


void medianFilter::setImage(float* array_host_)
{

//  double iStart = cpuSecond();

  array_host = array_host_;

  hipError_t err = hipMemcpy(array_device_in, array_host, insize, hipMemcpyHostToDevice);

//  printf("total copy to device time for this process took %f sec \n",(cpuSecond() - iStart));
  gpuErrchk( hipPeekAtLastError());
  assert(err == 0);

}

double medianFilter::cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

//void medianFilter::run3DRemoveOutliner(int size, int diff)
//{
//
//  double iStart = cpuSecond();
//
//  int block_size_x = BLOCK_X;
//  int block_size_y = BLOCK_Y;
//  int block_size_z = BLOCK_Z;
//
//  dim3 gridSize(((nx+block_size_x-1)/block_size_x), ((ny+block_size_y-1)/block_size_y), ((nz+block_size_z-1)/block_size_z));
//  dim3 blockSize(block_size_x, block_size_y, block_size_z);
//
//
//  switch(filterSize)
//  {
//    case 2:
//      reomveOutliner3D2<<<gridSize,blockSize>>>(nx, ny, nz, diff, array_device_out, array_device_in);
//      break;
////    case 3:
////      kernel3<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 4:
////      kernel4<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 5:
////      kernel5<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 6:
////      kernel6<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
//    case 15:
//      reomveOutliner3D15<<<gridSize,blockSize>>>(nx, ny, nz, diff, array_device_out, array_device_in);
//      break;
//    default:
//      break;
//
//  }
//
//
//  // add these to synchronzie the thread
////  hipDeviceSynchronize();
//
//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));
//
//  hipError_t err = hipGetLastError();
//  assert(err == 0);
//}


//void medianFilter::run3DFilter(int size)
//{
//
//  double iStart = cpuSecond();
//
//  int block_size_x = BLOCK_X;
//  int block_size_y = BLOCK_Y;
//  int block_size_z = BLOCK_Z;
//
//  dim3 gridSize(((nx+block_size_x-1)/block_size_x), ((ny+block_size_y-1)/block_size_y), ((nz+block_size_z-1)/block_size_z));
//  dim3 blockSize(block_size_x, block_size_y, block_size_z);
//
//  switch(filterSize)
//  {
//    case 2:
//      kernel3D2<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
////    case 3:
////      kernel3<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 4:
////      kernel4<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 5:
////      kernel5<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 6:
////      kernel6<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
//    case 15:
//      kernel3D15<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
//    default:
//      break;
//
//  }
//
//  gpuErrchk( hipPeekAtLastError());
//  gpuErrchk( hipDeviceSynchronize() );
//
//  // add these to synchronzie the thread
////  hipDeviceSynchronize();
//
//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));
//
////  hipError_t err = hipGetLastError();
////  assert(err == 0);
//}

//void medianFilter::run3DFilterXZ(int size)
//{
//
//  double iStart = cpuSecond();
//
//  int block_size_x = BLOCK_X;
//  int block_size_y = BLOCK_Y;
//  int block_size_z = BLOCK_Z;
//
//
//
//  int tmp =  nz;
//  nz = ny;
//  ny = tmp;
//
//  dim3 gridSize(((nx+block_size_x-1)/block_size_x), ((ny+block_size_y-1)/block_size_y), ((nz+block_size_z-1)/block_size_z));
//  dim3 blockSize(block_size_x, block_size_y, block_size_z);
//
//
//
//  switch(filterSize)
//  {
//    case 2:
//      kernel3D2<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
////    case 3:
////      kernel3<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 4:
////      kernel4<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 5:
////      kernel5<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 6:
////      kernel6<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
//    case 15:
////      kernel3D15XZ<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      kernel3D15XZME<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
//    default:
//      break;
//
//  }
//
//
//  gpuErrchk( hipPeekAtLastError());
//  gpuErrchk( hipDeviceSynchronize() );
//
//  // add these to synchronzie the thread
////  hipDeviceSynchronize();
//
//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));
//
////  hipError_t err = hipGetLastError();
////  assert(err == 0);
//}

//void medianFilter::run2DLoopFilter(int size)
//{
//
//  double iStart = cpuSecond();
//
//
//  int block_size_x = BLOCK_X;
//  int block_size_y = BLOCK_Y;
//
//  dim3 gridSize((nx+block_size_x-1)/block_size_x, (ny+block_size_y-1)/block_size_y);
//  dim3 blockSize(block_size_x,block_size_y);
//
//  switch(filterSize)
//  {
//    case 2:
//      kernel3D2<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
////    case 3:
////      kernel3<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 4:
////      kernel4<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 5:
////      kernel5<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 6:
////      kernel6<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
//    case 15:
//      kernelLool3D15<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
//    default:
//      break;
//
//  }
//  // add these to synchronzie the thread
//  hipDeviceSynchronize();
//
//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));
//
//  hipError_t err = hipGetLastError();
//  assert(err == 0);
//}

//void medianFilter::run2DLoopFilterXZY(int size)
//{
//
//  double iStart = cpuSecond();
//  int block_size_x = BLOCK_X;
//  int block_size_y = BLOCK_Y;
//
//  dim3 gridSize((nx+block_size_x-1)/block_size_x, (nz+block_size_y-1)/block_size_y);
//  dim3 blockSize(block_size_x,block_size_y);
//
//  switch(filterSize)
//  {
//    case 2:
//      kernel3D2<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
////    case 3:
////      kernel3<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 4:
////      kernel4<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 5:
////      kernel5<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
////    case 6:
////      kernel6<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
////      break;
//    case 15:
//      kernelLool3D15XZY<<<gridSize,blockSize>>>(nx, ny, nz, array_device_out, array_device_in);
//      break;
//    default:
//      break;
//
//  }
//  // add these to synchronzie the thread
//  hipDeviceSynchronize();
//
//  printf("total execution time for this kernel took %f sec \n",(cpuSecond() - iStart));
//
//  hipError_t err = hipGetLastError();
//  assert(err == 0);
//}
