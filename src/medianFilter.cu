#include <medianKernel.cu>
#include <medianFilter.hh>
#include <assert.h>
#include <iostream>

using namespace std;

medianFilter::medianFilter (float* array_host_, int nx_, int ny_, int filterSize_)
{

  array_host = array_host_;
  nx = nx_;
  ny = ny_;
  filterSize = filterSize_;



  inlength=(nx+filterSize-1)*(ny+filterSize-1);
  outlength = nx*ny;

//  for (int i =0; i< inlength; i++)
//    printf("the element is %f \n", array_host[i]);

  int insize = inlength * sizeof(float);
  int outsize = outlength * sizeof(float);

//  if ((array_host_out = (float*)malloc(sizeof(float)*length))) == 0)
//  {
//    fprintf(stderr,"malloc1 Fail \n");
//    return;
//  }

  hipError_t err = hipMalloc((void**) &array_device_in, insize);
  err = hipMalloc((void**) &array_device_out, outsize);

  assert(err == 0);
  err = hipMemcpy(array_device_in, array_host, insize, hipMemcpyHostToDevice);
  assert(err == 0);
}

void medianFilter::runFilter(int size)
{


//  kernel_add_one<<<64, 64>>>(array_device, length);

//  hipError_t err0 = hipMalloc((void**) &v_device, size*size*sizeof(float));
//  hipMemset(v_device, 0, size*size*sizeof(float));

  int block_size_x = BLOCK_X;
  int block_size_y = BLOCK_Y;

  dim3 blocks((nx+block_size_x-1)/block_size_x, (ny+block_size_y-1)/block_size_y);
  dim3 threads(block_size_x,block_size_y);

  switch(filterSize)
  {
    case 2:
      kernel2<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 3:
      kernel3<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 4:
      kernel4<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 5:
      kernel5<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 6:
      kernel6<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    case 15:
      kernel15<<<blocks,threads>>>(nx, ny, array_device_out, array_device_in);
      break;
    default:
      break;

  }

  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void medianFilter::retreive()
{



  int outsize = outlength * sizeof(float);
  hipMemcpy(array_host, array_device_out, outsize, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0)
  {
    cout << err << endl; assert(0);
  }

}

void medianFilter::retreive_to (float* array_host_)
{
//  assert(length == length_);

  int outsize = outlength * sizeof(float);
  hipMemcpy(array_host_, array_device_out, outsize, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

medianFilter::~medianFilter()
{

//  hipFree(v_device);

  hipFree(array_device_in);
  hipFree(array_device_out);

}
